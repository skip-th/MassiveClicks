#include "hip/hip_runtime.h"
/** CUDA utility functions.
 *
 * cuda_utils.cu:
 *  - Defines several utility functions to prevent code duplication.
 */

#include "cuda_utils.cuh"

// Source: Cuda Toolkit Documentation, B.14. Atomic Functions.
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
#if __CUDA_ARCH__ >= 600
    /**
     * @brief Atomically adds a float value to another float at a given
     * address for CUDA architecture >6.0.
     *
     * @param address The address to add the value to.
     * @param val The value to add.
     */
    DEV void atomicAddArch(float* address, const float val) {
        atomicAdd_system(address, val);
    }
#else
    /**
     * @brief Atomically adds a float value to another float at a given
     * address for CUDA architecture <6.0.
     *
     * @param address The address to add the value to.
     * @param val The value to add.
     */
    DEV void atomicAddArch(float* address, const float val) {
        unsigned int* address_as_ull = (unsigned int*) address;
        unsigned int old = *address_as_ull, assumed;

        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed,
                            __float_as_uint(val + __uint_as_float(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
        } while (assumed != old);
    }
#endif

/**
 * @brief Get the number of GPU devices available on this machine.
 *
 * @param num_devices Integer used to store the number of GPU devices.
 */
void get_number_devices(int *num_devices) {
    CUDA_CHECK(hipGetDeviceCount(num_devices));
}

/**
 * @brief Get the compute capability of a specific GPU device.
 *
 * @param device The ID of the GPU device.
 * @return int The compute capability of the GPU device (e.g. 5.2 = 52).
 */
int get_compute_capability(const int device) {
    hipDeviceProp_t dprop;
    CUDA_CHECK(hipGetDeviceProperties(&dprop, device));

    return std::stoi(std::to_string(dprop.major) + std::to_string(dprop.minor));
}

/**
 * @brief Get the memory usage of a specific GPU device.
 *
 * @param device_id The ID of the GPU device.
 * @param free_memory The free memory of the GPU device.
 * @param total_memory The total memory of the GPU device.
 * @param rounding The number by which the free and total memory will be rounded.
 */
void get_device_memory(const int& device_id, size_t& free_memory, size_t& total_memory, const size_t rounding) {
    int old_device;
    CUDA_CHECK(hipGetDevice(&old_device));
    CUDA_CHECK(hipSetDevice(device_id));

    size_t free_bytes, total_bytes;

    CUDA_CHECK(hipMemGetInfo(&free_bytes, &total_bytes));

    free_memory = free_bytes / rounding;
    total_memory = total_bytes / rounding;

    CUDA_CHECK(hipSetDevice(old_device));
}
