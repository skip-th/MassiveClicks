#include "hip/hip_runtime.h"
//
// Created by Pooya Khandel on 09/09/2021.
//

#include "factor.cuh"

/**
 * @brief Set the necessary arguments to compute phi.
 *
 * @param click_probs The current click probabilities for this SERP.
 * @param exam_probs The current examination probabilities for this SERP.
 * @param click The click on the current document.
 * @param last_click_rank The rank of the last clicked document in this SERP.
 * @param rank The rank of this document.
 * @param attr The attractiveness of this query-document pair.
 * @param tau_1 The first continuation parameter.
 * @param tau_2 The second continuation parameter.
 * @param tau_3 The third continuation parameter.
 */
DEV CCMFactor::CCMFactor(float (&click_probs)[MAX_SERP_LENGTH][MAX_SERP_LENGTH], float (&exam_probs)[MAX_SERP_LENGTH + 1], int click, int last_click_rank, int rank, float attr, float tau_1, float tau_2, float tau_3) {
    this->click_probs = click_probs;
    this->exam_probs = exam_probs;
    this->click = click;
    this->last_click_rank = last_click_rank;
    this->rank = rank;
    this->attr = attr;
    this->tau_1 = tau_1;
    this->tau_2 = tau_2;
    this->tau_3 = tau_3;
}

/**
 * @brief Compute the phi function.
 *
 * @param x The x input value for phi.
 * @param y The y input value for phi.
 * @param z The z input value for phi.
 */
DEV float CCMFactor::compute(int x, int y, int z, int qid, int did) {
    float log_prob = 0.f;
    // printf("%d, %d] (%d,%d,%d) [attr = %f, tau 1 = %f, tau 2 = %f, tau 3 = %f, last_click_rank = %d, rank = %d, click = %d\n", qid, did, x, y, z, this->attr, this->tau_1, this->tau_2, this->tau_3, this->last_click_rank, this->rank, this->click);
    // if (x == 1 && y == 0 && z == 1) {
    // }
    // int debug = 0;
    // if (x == 1 && y == 0 && z == 1) {
    //     debug = 1;
    // }
    // if (debug) {
        // for (int i = 0; i < MAX_SERP_LENGTH; i++) {
        //     for (int j = 0; j < MAX_SERP_LENGTH; j++) {
        //         printf("click_probs[%d][%d] = %f\n", i, j, this->click_probs[i][j]);
        //     }
        // }
        // for (int j = 0; j < MAX_SERP_LENGTH + 1; j++) {
        //     printf("exam_probs[%d] = %f\n", j, this->click_probs[j]);
        // }
    // }
    // if (debug) {printf("click = %d, last_click_rank = %d,  rank = %d, attr = %f, tau_1 = %f, tau_2 = %f, tau_3 = %f\n", this->click, this->last_click_rank, this->rank, this->attr, this->tau_1, this->tau_2, this->tau_3);}

    if (this->click == 0) { // Use tau 1 in case the document has not been clicked.
        if (y == 1) {
            // printf("%d, %d] (%d,%d,%d) 1, log_prob = %f, returned = %f, [attr = %f, tau 1 = %f, tau 2 = %f, tau 3 = %f, last_click_rank = %d, rank = %d, click = %d\n", qid, did, x, y, z, log_prob, 0.f, this->attr, this->tau_1, this->tau_2, this->tau_3, this->last_click_rank, this->rank, this->click);
            return 0.f;
        }

        log_prob += __logf(1 - this->attr);

        if (x == 1) {
            if (z == 1) {
                log_prob += __logf(this->tau_1);
                // if (debug) {printf("nc, tau1 = %f\n", this->tau_1);}
            }
            else {
                log_prob += __logf(1 - this->tau_1);
            }
        }
        else if (z == 1) {
            // printf("%d, %d] (%d,%d,%d) 2, log_prob = %f, returned = %f, [attr = %f, tau 1 = %f, tau 2 = %f, tau 3 = %f, last_click_rank = %d, rank = %d, click = %d\n", qid, did, x, y, z, log_prob, 0.f, this->attr, this->tau_1, this->tau_2, this->tau_3, this->last_click_rank, this->rank, this->click);
            return 0.f;
        }
    }
    else { // Use tau 2 or 3 in case the document has been clicked.
        if (x == 0) {
            // printf("%d, %d] (%d,%d,%d) 3, log_prob = %f, returned = %f, [attr = %f, tau 1 = %f, tau 2 = %f, tau 3 = %f, last_click_rank = %d, rank = %d, click = %d\n", qid, did, x, y, z, log_prob, 0.f, this->attr, this->tau_1, this->tau_2, this->tau_3, this->last_click_rank, this->rank, this->click);
            return 0.f;
        }

        log_prob += __logf(this->attr);

        if (y == 0) {
            log_prob += __logf(1 - this->attr);

            if (z == 1) {
                log_prob += __logf(this->tau_2);
            }
            else {
                log_prob += __logf(1 - this->tau_2);
            }
        }
        else {
            log_prob += __logf(this->attr);

            if (z == 1) {
                log_prob += __logf(this->tau_3);
            }
            else {
                log_prob += __logf(1 - this->tau_3);
            }

        }
    }

    if (z == 0) {
        if (this->last_click_rank >= (this->rank + 1)) {
            // printf("%d, %d] (%d,%d,%d) 4, log_prob = %f, returned = %f, [attr = %f, tau 1 = %f, tau 2 = %f, tau 3 = %f, last_click_rank = %d, rank = %d, click = %d\n", qid, did, x, y, z, log_prob, 0.f, this->attr, this->tau_1, this->tau_2, this->tau_3, this->last_click_rank, this->rank, this->click);
            return 0.f;
        }
    }
    else if (this->rank + 1 < MAX_SERP_LENGTH) {
        // for (int start_rank = this->rank + 1; start_rank < MAX_SERP_LENGTH; start_rank++) {
        //     int ses_itr{0};
        for (int res_itr = 0; res_itr < MAX_SERP_LENGTH - this->rank - 1; res_itr++) {
            log_prob += __logf(this->click_probs[this->rank + 1][res_itr]);
        }
        // }

        // for (int sub_rank = 0; sub_rank < MAX_SERP_LENGTH; sub_rank++) { // ! this part is most likely incorrect since print 5 always the wrong return value. replace this parclick with this snippet and change until it is correct.
        //     log_prob += __logf(this->click_probs[this->rank + 1][sub_rank]);
        // }
    }

    float exam_val = this->exam_probs[this->rank];

    if (x == 1) {
        log_prob += __logf(exam_val);
    }
    else {
        log_prob += __logf(1 - exam_val);
    }

    // printf("%d, %d] (%d,%d,%d) 5, log_prob = %f, returned = %f, [attr = %f, tau 1 = %f, tau 2 = %f, tau 3 = %f, last_click_rank = %d, rank = %d, click = %d\n", qid, did, x, y, z, log_prob, __expf(log_prob), this->attr, this->tau_1, this->tau_2, this->tau_3, this->last_click_rank, this->rank, this->click);
    return __expf(log_prob);
}

DEV DBNFactor::DBNFactor(float (&click_probs)[MAX_SERP_LENGTH][MAX_SERP_LENGTH], float (&exam_probs)[MAX_SERP_LENGTH + 1], int click,
                     int last_click_rank, int rank, float attr, float sat, float gamma) {
    this->click_probs = click_probs;
    this->exam_probs = exam_probs;
    this->click = click;
    this->last_click_rank = last_click_rank;
    this->rank = rank;
    this->attr = attr;
    this->gamma = gamma;
    this->sat = sat;
}

DEV float DBNFactor::compute(int x, int y, int z) {
    float log_prob = 0.f;

    if (this->click == 0){
        if (y == 1){
            return 0.f;
        }

        log_prob += __logf(1 - this->attr);

        if (x == 1) {
            if (z == 1) {
                log_prob += __logf(this->gamma);
            }
            else {
                log_prob += __logf(1 - this->gamma);
            }
        }
        else if (z == 1) {
            return 0.f;
        }
    }
    else {
        if (x == 0) {
            return 0.f;
        }

        log_prob += __logf(this->attr);

        if (y == 0){
            log_prob += __logf(1 - this->sat);
            if (z == 1) {
                log_prob += __logf(this->gamma);
            }
            else {
                log_prob += __logf(1 - this->gamma);
            }
        }
        else {
            if (z == 1) {
                return 0.f;
            }

            log_prob += __logf(this->sat);
        }
    }

    if (z == 0) {
        if (this->last_click_rank >= this->rank + 1) {
            return 0.f;
        }
    }
    else if (this->rank + 1 < MAX_SERP_LENGTH) {
        for (int sub_rank = 0; sub_rank < MAX_SERP_LENGTH; sub_rank++) {
            log_prob += __logf(this->click_probs[this->rank + 1][sub_rank]);
        }
    }

    float exam_val = this->exam_probs[this->rank];

    if (x == 1) {
        log_prob += __logf(exam_val);
    }
    else {
        log_prob += __logf(1 - exam_val);
    }

    return __expf(log_prob);
}