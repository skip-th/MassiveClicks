#include "hip/hip_runtime.h"
//
// Created by Pooya Khandel on 09/09/2021.
//

#include "factor.cuh"

DEV CCMFactor::CCMFactor(float (&click_probs)[MAX_SERP_LENGTH][MAX_SERP_LENGTH], float (&exam_probs)[MAX_SERP_LENGTH + 1], int click, int last_click_rank, int rank, float attr, float tau_1, float tau_2, float tau_3) {
    this->click_probs = click_probs;
    this->exam_probs = exam_probs;
    this->click = click;
    this->last_click_rank = last_click_rank;
    this->rank = rank;
    this->attr = attr;
    this->tau_1 = tau_1;
    this->tau_2 = tau_2;
    this->tau_3 = tau_3;
}

DEV float CCMFactor::compute(int x, int y, int z) {
    float log_prob = 0.f;

    if (this->click == 0) {
        if (y == 1) {
            return 0.f;
        }

        log_prob += __logf(1 - this->attr);

        if (x == 1) {
            if ( z == 1) {
                log_prob += __logf(this->tau_1);
            }
            else {
                log_prob += __logf(1 - this->tau_1);
            }
        }
        else if (z == 1) {
            return 0.f;
        }
    }
    else {
        if (x == 0) {
            return 0.f;
        }

        log_prob += __logf(this->attr);

        if (y == 0) {
            log_prob += __logf(1 - this->attr);
            if (z == 1) {
                log_prob += __logf(this->tau_2);
            }
            else {
                log_prob += __logf(1 - this->tau_2);
            }

        }
        else {
            log_prob += __logf(this->attr);

            if (z == 1) {
                log_prob += __logf(this->tau_3);
            }
            else {
                log_prob += __logf(1 - this->tau_3);
            }

        }
    }

    if (z == 0) {
        if (this->last_click_rank >= (this->rank + 1)) {
            return 0.f;
        }
    }
    else if (this->rank + 1 < MAX_SERP_LENGTH) {
        // log_prob += std::accumulate(tail_clicks.begin(), tail_clicks.end(), 0.0, [](const double base, decltype(*begin(tail_clicks))& element){return base + std::log(element.second);});
        for (int sub_rank = 0; sub_rank < MAX_SERP_LENGTH; sub_rank++) {
            log_prob += __logf(this->click_probs[this->rank + 1][sub_rank]);
        }
    }

    float exam_val = this->exam_probs[this->rank];

    if (x == 1) {
        log_prob += __logf(exam_val);
    }
    else {
        log_prob += __logf(1 - exam_val);
    }

    return __expf(log_prob);
}

DEV DBNFactor::DBNFactor(float (&click_probs)[MAX_SERP_LENGTH][MAX_SERP_LENGTH], float (&exam_probs)[MAX_SERP_LENGTH + 1], int click,
                     int last_click_rank, int rank, float attr, float sat, float gamma) {
    this->click_probs = click_probs;
    this->exam_probs = exam_probs;
    this->click = click;
    this->last_click_rank = last_click_rank;
    this->rank = rank;
    this->attr = attr;
    this->gamma = gamma;
    this->sat = sat;
}

DEV float DBNFactor::compute(int x, int y, int z) {
    float log_prob = 0.f;

    if (this->click == 0){
        if (y == 1){
            return 0.f;
        }

        log_prob += __logf(1 - this->attr);

        if (x == 1) {
            if (z == 1) {
                log_prob += __logf(this->gamma);
            }
            else {
                log_prob += __logf(1 - this->gamma);
            }
        }
        else if (z == 1) {
            return 0.f;
        }
    }
    else {
        if (x == 0) {
            return 0.f;
        }

        log_prob += __logf(this->attr);

        if (y == 0){
            log_prob += __logf(1 - this->sat);
            if (z == 1) {
                log_prob += __logf(this->gamma);
            }
            else {
                log_prob += __logf(1 - this->gamma);
            }
        }
        else {
            if (z == 1) {
                return 0.f;
            }

            log_prob += __logf(this->sat);
        }
    }

    if (z == 0) {
        if (this->last_click_rank >= this->rank + 1) {
            return 0.f;
        }
    }
    else if (this->rank + 1 < MAX_SERP_LENGTH) {
        for (int sub_rank = 0; sub_rank < MAX_SERP_LENGTH; sub_rank++) {
            log_prob += __logf(this->click_probs[this->rank + 1][sub_rank]);
        }
    }

    float exam_val = this->exam_probs[this->rank];

    if (x == 1) {
        log_prob += __logf(exam_val);
    }
    else {
        log_prob += __logf(1 - exam_val);
    }

    return __expf(log_prob);
}