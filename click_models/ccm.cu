#include "hip/hip_runtime.h"
/** First implementation of a CCM.
 * Pooya Khandel's ParClick is used as a reference implementation.
 *
 * ccm.cu:
 *  - Defines the functions specific to creating a CCM CM.
 */

#include "ccm.cuh"


//---------------------------------------------------------------------------//
// Host-side CCM click model functions.                                      //
//---------------------------------------------------------------------------//

HST CCM_Host::CCM_Host() = default;

/**
 * @brief Constructs a CCM click model object for the host.
 *
 * @param ccm
 * @returns CCM_Host The CCM click model object.
 */
HST CCM_Host::CCM_Host(CCM_Host const &ccm) {
}

/**
 * @brief Creates a new CCM click model object.
 *
 * @return CCM_Host* The CCM click model object.
 */
HST CCM_Host* CCM_Host::clone() {
    return new CCM_Host(*this);
}

/**
 * @brief Print a message.
 */
HST void CCM_Host::say_hello() {
    std::cout << "Host-side CCM says hello!" << std::endl;
}

// /**
//  * @brief Get the click probability of a search result.
//  *
//  * @param qd_parameter_index The query-document pair parameter index of the
//  * search result.
//  * @param rank The document rank of the search result.
//  * @return float The click probability.
//  */
// HST float CCM_Host::get_click_probability(int& qd_parameter_index, int& rank) {
//     return this->attractiveness_parameters[qd_parameter_index].value() * this->examination_parameters[rank].value();
// }

/**
 * @brief Get the amount of device memory allocated to this click model.
 *
 * @return size_t The used memory.
 */
HST size_t CCM_Host::get_memory_usage(void) {
    return this->cm_memory_usage;
}

/**
 * @brief Allocate device-side memory for the attractiveness parameters.
 *
 * @param partition The training and testing sets, and the number of
 * query-document pairs in the training set.
 * @param n_devices The number of devices on this node.
 */
HST void CCM_Host::init_attractiveness_parameters(const std::tuple<std::vector<SERP>, std::vector<SERP>, int>& partition, int n_devices) {
    Param default_parameter;
    default_parameter.set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);

    // Allocate memory for the attractiveness parameters on the device.
    this->n_attr_dev = std::get<2>(partition);
    this->attractiveness_parameters.resize(this->n_attr_dev, default_parameter);
    CUDA_CHECK(hipMalloc(&this->attr_param_dptr, this->n_attr_dev * sizeof(Param)));
    CUDA_CHECK(hipMemcpy(this->attr_param_dptr, this->attractiveness_parameters.data(),
                          this->n_attr_dev * sizeof(Param), hipMemcpyHostToDevice));

    // Allocate memory for the temporary attractiveness parameters on the device.
    // These values are replaced at the start of each iteration, which means
    // they don't need to be initialized with a CUDA memory copy.
    // this->n_tmp_attr_dev = std::get<0>(partition).size() * MAX_SERP_LENGTH;
    // this->tmp_attractiveness_parameters.resize(this->n_tmp_attr_dev);
    // CUDA_CHECK(hipMalloc(&this->tmp_attr_param_dptr, this->n_tmp_attr_dev * sizeof(Param)));
    this->n_tmp_attr_dev = std::get<0>(partition).size() * MAX_SERP_LENGTH;
    this->tmp_attractiveness_parameters.resize(this->n_tmp_attr_dev, default_parameter);
    CUDA_CHECK(hipMalloc(&this->tmp_attr_param_dptr, this->n_tmp_attr_dev * sizeof(Param)));
    // CUDA_CHECK(hipMemcpy(this->tmp_attr_param_dptr, this->tmp_attractiveness_parameters.data(),
    //                       this->n_tmp_attr_dev * sizeof(Param), hipMemcpyHostToDevice));

    // Store the number of allocated bytes.
    this->cm_memory_usage += this->n_attr_dev * sizeof(Param) + this->n_tmp_attr_dev * sizeof(Param);
}

/**
 * @brief Allocate device-side memory for the continuation parameters, tau.
 *
 * @param partition The training and testing sets, and the number of
 * query-document pairs in the training set.
 * @param n_devices The number of devices on this node.
 */
HST void CCM_Host::init_tau_parameters(const std::tuple<std::vector<SERP>, std::vector<SERP>, int>& partition, int n_devices) {
    Param default_parameter;
    default_parameter.set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);

    // Allocate memory for the continuation parameters on the device.
    this->n_tau_dev = 3;
    this->tau_parameters.resize(this->n_tau_dev, default_parameter);
    CUDA_CHECK(hipMalloc(&this->tau_param_dptr, this->n_tau_dev * sizeof(Param)));
    CUDA_CHECK(hipMemcpy(this->tau_param_dptr, this->tau_parameters.data(),
                          this->n_tau_dev * sizeof(Param), hipMemcpyHostToDevice));

    // Allocate memory for the temporary continuation parameters on the device.
    // These values are replaced at the start of each iteration, which means
    // they don't need to be initialized with a CUDA memory copy.
    this->n_tmp_tau_dev = std::get<0>(partition).size() * this->n_tau_dev;
    this->tmp_tau_parameters.resize(this->n_tmp_tau_dev);
    CUDA_CHECK(hipMalloc(&this->tmp_tau_param_dptr, this->n_tmp_tau_dev * sizeof(Param)));
    // CUDA_CHECK(hipMemcpy(this->tmp_tau_param_dptr, this->tmp_tau_parameters.data(),
    //                       this->n_tmp_tau_dev * sizeof(Param), hipMemcpyHostToDevice));

    // Store the number of allocated bytes.
    this->cm_memory_usage += this->n_tau_dev * sizeof(Param) + this->n_tmp_tau_dev * sizeof(Param);
}

/**
 * @brief Allocate device-side memory for the attractiveness and continuation
 * parameters of the click model.
 *
 * @param partition The training and testing sets, and the number of
 * query-document pairs in the training set.
 * @param n_devices The number of devices on this node.
 */
HST void CCM_Host::init_parameters(const std::tuple<std::vector<SERP>, std::vector<SERP>, int>& partition, int n_devices) {
    this->init_attractiveness_parameters(partition, n_devices);
    this->init_tau_parameters(partition, n_devices);
}

/**
 * @brief Get the references to the allocated device-side memory.
 *
 * @param param_refs An array containing the references to the device-side
 * parameters in memory.
 * @param param_sizes The size of each of the memory allocations on the device.
 */
HST void CCM_Host::get_device_references(Param**& param_refs, int*& param_sizes) {
    int n_references = 4;

    // Create a temporary array to store the device references.
    Param* tmp_param_refs_array[n_references];
    tmp_param_refs_array[0] = this->attr_param_dptr;
    tmp_param_refs_array[1] = this->tmp_attr_param_dptr;
    tmp_param_refs_array[2] = this->tau_param_dptr;
    tmp_param_refs_array[3] = this->tmp_tau_param_dptr;

    // Allocate space for the device references.
    CUDA_CHECK(hipMalloc(&param_refs, n_references * sizeof(Param*)));
    CUDA_CHECK(hipMemcpy(param_refs, tmp_param_refs_array,
                          n_references * sizeof(Param*), hipMemcpyHostToDevice));

    int tmp_param_sizes_array[n_references];
    tmp_param_sizes_array[0] = this->n_attr_dev;
    tmp_param_sizes_array[1] = this->n_tmp_attr_dev;
    tmp_param_sizes_array[2] = this->n_tau_dev;
    tmp_param_sizes_array[3] = this->n_tmp_tau_dev;

    // Allocate space for the device references.
    CUDA_CHECK(hipMalloc(&param_sizes, n_references * sizeof(int)));
    CUDA_CHECK(hipMemcpy(param_sizes, tmp_param_sizes_array,
                          n_references * sizeof(int), hipMemcpyHostToDevice));

    // Keep track of the pointers to the allocated device-side memory.
    this->param_refs = param_refs;
    this->param_sizes = param_sizes;
    this->cm_memory_usage += n_references * sizeof(Param*) + n_references * sizeof(int);
}

/**
 * @brief Update the global parameter values with the temporarily stored new
 * local parameter values on each thread.
 *
 * @param gridSize The size of kernel blocks on the GPU.
 * @param blockSize The number of kernel threads per block on the GPU.
 * @param partition The dataset allocated on the GPU.
 * @param dataset_size The size of the allocated dataset.
 */
HST void CCM_Host::update_parameters(int& gridSize, int& blockSize, SERP*& partition, int& dataset_size) {
    Kernel::update<<<gridSize, blockSize>>>(partition, dataset_size, 0);
}

/**
 * @brief Reset the original parameter values to zero so the previous parameter
 * values won't affect the next result twice.
 *
 * Further explanation; The first time it would affect the result would be when
 * retrieving their values in the training kernel, and the (unnecessary) second
 * time would be when adding the values to the original parameter containers.
 * The second time would still give a valid result but would slow down the
 * converging of the parameters.
 */
HST void CCM_Host::reset_parameters(void) {
    // Create a parameter initialized at 0.
    Param default_parameter;
    default_parameter.set_values(1.f, 2.f);

    // Create an array of the right proportions with the empty parameters.
    std::vector<Param> cleared_attractiveness_parameters(this->n_attr_dev, default_parameter);
    std::vector<Param> cleared_tau_parameters(this->n_attr_dev, default_parameter);
    // std::vector<Param> cleared_tmp_attractiveness_parameters(this->n_tmp_attr_dev, default_parameter);
    // std::vector<Param> cleared_tmp_tau_parameters(this->n_tmp_tau_dev, default_parameter);

    // Copy the cleared array to the device.
    CUDA_CHECK(hipMemcpy(this->attr_param_dptr, cleared_attractiveness_parameters.data(), this->n_attr_dev * sizeof(Param), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(this->tau_param_dptr, cleared_tau_parameters.data(), this->n_tau_dev * sizeof(Param), hipMemcpyHostToDevice));
    // CUDA_CHECK(hipMemcpy(this->tmp_attr_param_dptr, cleared_tmp_attractiveness_parameters.data(), this->n_tmp_attr_dev * sizeof(Param), hipMemcpyHostToDevice));
    // CUDA_CHECK(hipMemcpy(this->tmp_tau_param_dptr, cleared_tmp_tau_parameters.data(), this->n_tmp_tau_dev * sizeof(Param), hipMemcpyHostToDevice));
}

/**
 * @brief Transfers parameters of a given type either from the device to the
 * host, or the otherway around.
 *
 * @param parameter_type The type of parameter that will be transfered.
 * (PUBLIC, PRIVATE, or ALL).
 * @param transfer_direction The direction in which the transfer will happen.
 * (H2D or D2H).
 */
HST void CCM_Host::transfer_parameters(int parameter_type, int transfer_direction) {
    // Public parameters.
    if (parameter_type == PUBLIC || parameter_type == ALL) {
        if (transfer_direction == D2H) { // Transfer from device to host.
            // Retrieve the continuation parameters from the device.
            CUDA_CHECK(hipMemcpy(this->tau_parameters.data(), this->tau_param_dptr, this->n_tau_dev * sizeof(Param), hipMemcpyDeviceToHost));
        }
        else if (transfer_direction == H2D) { // Transfer from host to device.
            // Send the continuation parameters to the device.
            CUDA_CHECK(hipMemcpy(this->tau_param_dptr, this->tau_parameters.data(), this->n_tau_dev * sizeof(Param), hipMemcpyHostToDevice));
        }
    }

    // Private parameters.
    if (parameter_type == PRIVATE || parameter_type == ALL) {
        if (transfer_direction == D2H) { // Transfer from device to host.
            // Retrieve the attractiveness parameters from the device.
            CUDA_CHECK(hipMemcpy(this->attractiveness_parameters.data(), this->attr_param_dptr, this->n_attr_dev * sizeof(Param), hipMemcpyDeviceToHost));
        }
        else if (transfer_direction == H2D) { // Transfer from host to device.
            // Send the attractiveness parameters to the device.
            CUDA_CHECK(hipMemcpy(this->attr_param_dptr, this->attractiveness_parameters.data(), this->n_attr_dev * sizeof(Param), hipMemcpyHostToDevice));
        }
    }
}

/**
 * @brief Retrieve the parameters of a given type into a given array from the
 * click model.
 *
 * @param destination The array which will save the indicated parameters.
 * @param parameter_type The type of parameters which will be retrieved
 * (PUBLIC, PRIVATE, or ALL).
 */
HST void CCM_Host::get_parameters(std::vector<std::vector<Param>>& destination, int parameter_type) {
    // Add the parameters to a generic vector which can represent  multiple
    // retrieved parameter types.
    if (parameter_type == PUBLIC) {
        destination.resize(1);
        destination[0] = this->tau_parameters;
    }
    else if (parameter_type == PRIVATE) {
        destination.resize(1);
        destination[0] = this->attractiveness_parameters;
    }
    else if (parameter_type == ALL) {
        destination.resize(2);
        destination[0] = this->attractiveness_parameters;
        destination[1] = this->tau_parameters;
    }
}

/**
 * @brief Compute the result of combining the CCM parameters from other nodes
 * or devices.
 *
 * @param parameters A multi-dimensional vector containing the parameters to be
 * combined. The vector is structured as follows: Node/Device ID -> Parameter
 * type -> Parameters.
 */
HST void CCM_Host::sync_parameters(std::vector<std::vector<std::vector<Param>>>& parameters) {
    for (int r = 0; r < parameters[0][0].size(); r++) {
        for (int param_type = 0; param_type < parameters[0].size(); param_type++) {
            Param base = parameters[0][param_type][r];

            // Subtract the starting values of other partitions.
            parameters[0][param_type][r].set_values(base.numerator_val() - (parameters.size() - 1),
                                                    base.denominator_val() - 2 * (parameters.size() - 1));

            for (int device_id = 1; device_id < parameters.size(); device_id++) {
                Param ex = parameters[device_id][param_type][r];
                parameters[0][param_type][r].add_to_values(ex.numerator_val(),
                                                           ex.denominator_val());
            }
        }
    }
}

/**
 * @brief Set the parameters of a host-side click model equal to the given
 * given arguments.
 *
 * @param source The new parameter values.
 * @param parameter_type The type of the given parameters. (PUBLIC, PRIVATE, or
 * ALL).
 */
HST void CCM_Host::set_parameters(std::vector<std::vector<Param>>& source, int parameter_type) {
    // Set the parameters of this click model.
    if (parameter_type == PUBLIC) {
        this->tau_parameters = source[0];
    }
    else if (parameter_type == PRIVATE) {
        this->attractiveness_parameters = source[0];
    }
    else if (parameter_type == ALL) {
        this->attractiveness_parameters = source[0];
        this->tau_parameters = source[1];
    }
}

/**
 * @brief Compute the log-likelihood of the current CCM for the given query
 * session.
 *
 * @param query_session The query session for which the log-likelihood will be
 * computed.
 * @param log_click_probs The vector which will store the log-likelihood for
 * the document at each rank in the query session.
 */
HST void CCM_Host::get_log_conditional_click_probs(SERP& query_session, std::vector<float>& log_click_probs) {
    float atr, tau_1, tau_2, tau_3;
    float ex{1.f}, click_prob;

    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        SearchResult sr = query_session[rank];

        atr = (float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM;
        if (sr.get_param_index() != -1)
            atr = this->attractiveness_parameters[sr.get_param_index()].value();
        tau_1 = this->tau_parameters[0].value();
        tau_2 = this->tau_parameters[1].value();
        tau_3 = this->tau_parameters[2].value();

        if (sr.get_click() == 1) {
            click_prob = atr * ex;
            ex = tau_2 * (1 - atr) + tau_3 * atr;
        }
        else {
            click_prob = 1 - atr * ex;
            ex *= tau_1 * (1 - atr) / click_prob;
        }
        // printf("%d, %d] atr = %f, ex = %f, tau 1 = %f, tau 2 = %f, tau 3 = %f, click_prob = %f\n",
        //     query_session.get_query(), sr.get_doc_id(), atr, ex, tau_1, tau_2, tau_3, std::log(click_prob));

        log_click_probs.push_back(std::log(click_prob));
    }
}

/**
 * @brief Compute the click probability of the current CCM for the given query
 * session.
 *
 * @param query_session The query session for which the click probability will
 * be computed.
 * @param full_click_probs The vector which will store the click probability
 * for the document at each rank in the query session.
 */
HST void CCM_Host::get_full_click_probs(SERP& query_session, std::vector<float> &full_click_probs) {
    float atr, tau_1, tau_2, tau_3;
    float ex{1.f}, atr_mul_ex;

    // Go through all ranks of the query session.
    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        // Retrieve the search result at the current rank.
        SearchResult sr = query_session[rank];

        atr = (float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM;
        if (sr.get_param_index() != -1)
            atr = this->attractiveness_parameters[sr.get_param_index()].value();
        tau_1 = this->tau_parameters[0].value();
        tau_2 = this->tau_parameters[1].value();
        tau_3 = this->tau_parameters[2].value();

        // Calculate the click probability.
        atr_mul_ex = atr * ex;
        // printf("%d, %d] atr = %f, ex = %f, tau 1 = %f, tau 2 = %f, tau 3 = %f, atr_mul_ex = %f\n",
        //     query_session.get_query(), sr.get_doc_id(), atr, ex, tau_1, tau_2, tau_3, atr_mul_ex);

        // Calculate the full click probability.
        int click{sr.get_click()};
        if (click == 1) {
            full_click_probs.push_back(atr_mul_ex);
        }
        else {
            full_click_probs.push_back(1 - atr_mul_ex);
        }

        ex *= (1 - atr) * tau_1 + atr * ((1 - atr) * tau_2 + atr * tau_3);
    }
}

/**
 * @brief Frees the memory allocated to the parameters of this click model on
 * the GPU device.
 */
HST void CCM_Host::destroy_parameters(void) {
    // Free origin and temporary attractiveness containers.
    CUDA_CHECK(hipFree(this->attr_param_dptr));
    CUDA_CHECK(hipFree(this->tmp_attr_param_dptr));

    // Free origin and temporary continuation containers.
    CUDA_CHECK(hipFree(this->tau_param_dptr));
    CUDA_CHECK(hipFree(this->tmp_tau_param_dptr));

    // Free the device parameter references and sizes.
    CUDA_CHECK(hipFree(this->param_refs));
    CUDA_CHECK(hipFree(this->param_sizes));

    // Reset used device memory.
    this->cm_memory_usage = 0;
}


//---------------------------------------------------------------------------//
// Device-side CCM click model functions.                                    //
//---------------------------------------------------------------------------//

/**
 * @brief Prints a message.
 */
DEV void CCM_Dev::say_hello() {
    printf("Device-side CCM says hello!\n");
}

/**
 * @brief Creates a new CCM click model object.
 *
 * @return CCM_Dev* The CCM click model object.
 */
DEV CCM_Dev *CCM_Dev::clone() {
    return new CCM_Dev(*this);
}

DEV CCM_Dev::CCM_Dev() = default;

/**
 * @brief Constructs a CCM click model object for the device.
 *
 * @param ccm
 * @returns CCM_Dev The CCM click model object.
 */
DEV CCM_Dev::CCM_Dev(CCM_Dev const &ccm) {
}

/**
 * @brief Set the location of the memory allocated for the parameters of this
 * click model on the GPU device.
 *
 * @param parameter_ptr The pointers to the allocated memory.
 * @param parameter_sizes The size of the allocated memory.
 */
DEV void CCM_Dev::set_parameters(Param**& parameter_ptr, int* parameter_sizes) {
    // Set pointers to parameter arrays.
    this->attractiveness_parameters = parameter_ptr[0];
    this->tmp_attractiveness_parameters = parameter_ptr[1];
    this->tau_parameters = parameter_ptr[2];
    this->tmp_tau_parameters = parameter_ptr[3];

    // Set parameter array sizes.
    this->n_attractiveness_parameters = parameter_sizes[0];
    this->n_tmp_attractiveness_parameters = parameter_sizes[1];
    this->n_tau_parameters = parameter_sizes[2];
    this->n_tmp_tau_parameters = parameter_sizes[3];
}

/**
 * @brief Compute a single Expectation-Maximization iteration for the CCM click
 * model, for a single query session.
 *
 * @param query_session The query session which will be used to estimate the
 * CCM parameters.
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 */
DEV void CCM_Dev::process_session(SERP& query_session, int& thread_index) {
    // for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
    //     printf("%d, %d] attr = %f / %f = %f\n", query_session.get_query(), query_session[rank].get_doc_id(),
    //         this->attractiveness_parameters[query_session[rank].get_param_index()].numerator_val(),
    //         this->attractiveness_parameters[query_session[rank].get_param_index()].denominator_val(),
    //         this->attractiveness_parameters[query_session[rank].get_param_index()].value());
    // }
    int last_click_rank = query_session.last_click_rank();
    float click_probs[MAX_SERP_LENGTH][MAX_SERP_LENGTH] = { 0.f };
    float exam_probs[MAX_SERP_LENGTH + 1];
    float exam[MAX_SERP_LENGTH + 1];
    float car[MAX_SERP_LENGTH + 1] = { 0.f };

    // this->tmp_tau_parameters[thread_index * 3 + 0].set_values(this->tau_parameters[0].numerator_val(), this->tau_parameters[0].denominator_val());
    // this->tmp_tau_parameters[thread_index * 3 + 1].set_values(this->tau_parameters[1].numerator_val(), this->tau_parameters[1].denominator_val());
    // this->tmp_tau_parameters[thread_index * 3 + 2].set_values(this->tau_parameters[2].numerator_val(), this->tau_parameters[2].denominator_val());
    // this->tmp_tau_parameters[thread_index * 3 + 0].set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);
    // this->tmp_tau_parameters[thread_index * 3 + 1].set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);
    // this->tmp_tau_parameters[thread_index * 3 + 2].set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);
    this->tmp_tau_parameters[thread_index * 3 + 0].set_values(0.f, 0.f);
    this->tmp_tau_parameters[thread_index * 3 + 1].set_values(0.f, 0.f);
    this->tmp_tau_parameters[thread_index * 3 + 2].set_values(0.f, 0.f);

    this->compute_exam_car(thread_index, query_session, exam, car);
    this->compute_ccm_attr(thread_index, query_session, last_click_rank, exam, car);
    this->get_tail_clicks(thread_index, query_session, click_probs, exam_probs);
    this->compute_taus(thread_index, query_session, last_click_rank, click_probs, exam_probs);

    // ! Check the number of sessions assigned to the GPU (does it overlap somewhere?). There seem to be a quite a few sessions missing when looking at the tau.
    // ! The number of missing sessions is random which probably also causes the random results.

    // for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
    //     float click_probs_sum = 0.f;
    //     for (int srank = 0; srank < MAX_SERP_LENGTH - rank - 1; srank++) {
    //         click_probs_sum += click_probs[rank][srank];
    //     }
    //     printf("%d, %d] lcr = %d, atr = %f, exam[%d] = %f, exam_probs[%d] = %f, car[%d] = %f, click_probs = %f\n",
    //         query_session.get_query(), query_session[rank].get_doc_id(), last_click_rank, this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].value(), rank, exam[rank], rank, exam_probs[rank], rank, car[rank], click_probs_sum);
    // }

    // printf("%d\n", query_session.get_query());

    // printf("%d] last_click_rank = %d\n", query_session.get_query(), last_click_rank);
    // for (int i = 0; i < MAX_SERP_LENGTH + 1; i++) {
    //     printf("%d] exam[%d] = %f\n", query_session.get_query(), i, exam[i]);
    // }
    // for (int i = 0; i < MAX_SERP_LENGTH + 1; i++) {
    //     printf("%d] exam_probs[%d] = %f\n", query_session.get_query(), i, exam_probs[i]);
    // }
    // for (int i = 0; i < MAX_SERP_LENGTH + 1; i++) {
    //     printf("%d] car[%d] = %f\n", query_session.get_query(), i, car[i]);
    // }
    // for (int i = 0; i < MAX_SERP_LENGTH; i++) {
    //     for (int j = 0; j < MAX_SERP_LENGTH - j - 1; j++) {
    //         printf("%d, %d] click_probs[%d][%d] = %f\n", query_session.get_query(), query_session[i].get_doc_id(), i, j, click_probs[i][j]);
    //     }
    // }
    // for (int i = 0; i < 3; i++) {
    //     printf("%d] old tau[%d] = %f\n", query_session.get_query(), i, this->tau_parameters[i].value());
    // }

    // for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
    //     for (int i = 0; i < 3; i++) {
    //         printf("%d, %d] new tau[%d] = %f\n", query_session.get_query(), query_session[rank].get_doc_id(), i, this->tmp_tau_parameters[thread_index * 3 + i].value());
    //     }
    // }
}

DEV void CCM_Dev::compute_exam_car(int& thread_index, SERP& query_session, float (&exam)[MAX_SERP_LENGTH + 1], float (&car)[MAX_SERP_LENGTH + 1]) {
    // Set the default examination value for the first rank.
    exam[0] = 1.f;

    float attr_val, tau_1, tau_2, tau_3, ex_value, temp, car_val;

    float car_helper[MAX_SERP_LENGTH][2];

    for (int rank = 0; rank < MAX_SERP_LENGTH;) {
        SearchResult sr = query_session[rank];

        attr_val = this->attractiveness_parameters[sr.get_param_index()].value();
        tau_1 = this->tau_parameters[0].value();
        tau_2 = this->tau_parameters[1].value();
        tau_3 = this->tau_parameters[2].value();
        ex_value = exam[rank];

        temp = (1 - attr_val) * tau_1;

        // Calculate epsilon for the next rank.
        ex_value *= temp + attr_val * ((1 - attr_val) * tau_2 + attr_val * tau_3);

        car_helper[rank][0] = attr_val;
        car_helper[rank][1] = temp;

        // if (query_session.get_query() == 1421 && sr.get_doc_id() == 12596) {
        //     // printf("1) attr = %f, ex = %f,  exam[%d] = %f, tau 1 = %f, tau 2 = %f, tau 3 = %f, temp = %f\n", attr_val, ex_value, rank, exam[rank], tau_1, tau_2, tau_3, temp);
        //     printf("1) car_helper[%d][1] = %f = (1 - %f) * %f \n", rank, temp, attr_val, tau_1);
        // }

        // Set the examination value for the next rank.
        rank += 1;
        exam[rank] = ex_value;
    }

    // car = {0};
    for (int car_itr = MAX_SERP_LENGTH - 1; car_itr > -1; car_itr--) {
        car_val = car[car_itr + 1];

        // if (query_session.get_query() == 1421 && query_session[car_itr].get_doc_id() == 12596) {
        //     printf("2) %f = car[%d], %f = car[%d + 1], car[%d] = %f + %f * %f = %f\n",
        //         car[car_itr], car_itr,
        //         car[car_itr + 1], car_itr,
        //         car_itr, car_helper[car_itr][0], car_helper[car_itr][1], car_val, car_helper[car_itr][0] + car_helper[car_itr][1] * car_val);
        // }

        car[car_itr] = car_helper[car_itr][0] + car_helper[car_itr][1] * car_val;
    }
}

DEV void CCM_Dev::compute_ccm_attr(int& thread_index, SERP& query_session, int& last_click_rank, float (&exam)[MAX_SERP_LENGTH + 1], float (&car)[MAX_SERP_LENGTH + 1]) {
    float numerator_update, denominator_update;
    float attr_val, exam_val, car_val;

    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        SearchResult sr = query_session[rank];
        int click = sr.get_click();

        // int printed = 0;
        // printf("%d, %d] exam[%d] = %f\n", query_session.get_query(), sr.get_doc_id(), rank, exam[rank]);
        // printf("%d, %d] car[%d] = %f, car[%d + 1] = %f\n", query_session.get_query(), sr.get_doc_id(), rank, car[rank], rank, car[rank + 1]);

        numerator_update = 0.f;
        denominator_update = 1.f;

        attr_val = this->attractiveness_parameters[sr.get_param_index()].value();
        exam_val = exam[rank];

        if (click == 1) {
            numerator_update += 1;
            denominator_update += 1;
        }
        else if (rank >= last_click_rank) {
            car_val = car[rank];
            numerator_update += ((1 - exam_val) * attr_val) / (1 - exam_val * car_val);

            // if (query_session.get_query() == 1421 && sr.get_doc_id() == 12596) {
            //     printed = 1;
            //     printf("%d, %d] click = %d, attr = %f, exam = %f, car[ %d ](%d == 1 && %d >= %d || %d == 1 && %d == %d) = %f, new attr = %f / %f = %f\n",
            //         query_session.get_query(), sr.get_doc_id(), click, attr_val, exam_val, rank, click, rank, last_click_rank, click, rank, last_click_rank,
            //         car_val, numerator_update, denominator_update, numerator_update/denominator_update);
            // }
        }

        if (click == 1 && rank == last_click_rank) {
            car_val = car[rank + 1];
            numerator_update += attr_val / (1 - (this->tau_parameters[1].value() * (1 - attr_val) + this->tau_parameters[2].value() * attr_val) * car_val);

            // if (query_session.get_query() == 1421 && sr.get_doc_id() == 12596) {
            //     printed = 1;
            //     printf("%d, %d] click = %d, attr = %f, exam = %f, car[%d + 1](%d == 1 && %d >= %d || %d == 1 && %d == %d) = %f, new attr = %f / %f = %f\n",
            //         query_session.get_query(), sr.get_doc_id(), click, attr_val, exam_val, rank, click, rank, last_click_rank, click, rank, last_click_rank,
            //         car_val, numerator_update, denominator_update, numerator_update/denominator_update);
            // }
        }

        this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].set_values(numerator_update, denominator_update);


        // if (query_session.get_query() == 1421 && sr.get_doc_id() == 12596 && printed == 0) {
        //     printed = 1;
        //     printf("%d, %d] click = %d, attr = %f, exam = %f, car[def](%d == 1 && %d >= %d || %d == 1 && %d == %d) = %f, new attr = %f / %f = %f\n",
        //         query_session.get_query(), sr.get_doc_id(), click, attr_val, exam_val, click, rank, last_click_rank, click, rank, last_click_rank,
        //         car_val, numerator_update, denominator_update, numerator_update/denominator_update);
        // }

        // printf("%d, %d] attr = %f / %f = %f ?= (%f / %f = %f)\n", query_session.get_query(), sr.get_doc_id(),
        //     numerator_update, denominator_update, numerator_update/denominator_update,
        //     this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].numerator_val(), this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].denominator_val(), this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].value());
        // this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].add_to_values(numerator_update, denominator_update);
    }
}

DEV void CCM_Dev::get_tail_clicks(int& thread_index, SERP& query_session, float (&click_probs)[MAX_SERP_LENGTH][MAX_SERP_LENGTH], float (&exam_probs)[MAX_SERP_LENGTH + 1]) {
    exam_probs[0] = 1.f;
    float tau_1, tau_2, tau_3;
    float exam_val, click_prob;

    for (int start_rank = 0; start_rank < MAX_SERP_LENGTH; start_rank++) {
        exam_val = 1.f;

        int ses_itr{0};
        for (int res_itr = start_rank; res_itr < MAX_SERP_LENGTH; res_itr++) {
            SearchResult tmp_sr = query_session[ses_itr];

            float attr_val = this->attractiveness_parameters[tmp_sr.get_param_index()].value();
            tau_1 = this->tau_parameters[0].value();
            tau_2 = this->tau_parameters[1].value();
            tau_3 = this->tau_parameters[2].value();

            if (query_session[res_itr].get_click() == 1) {
                click_prob = attr_val * exam_val;
                exam_val = tau_2 * (1 - attr_val) + tau_3 * attr_val;
            }
            else {
                click_prob = 1 - attr_val * exam_val;
                exam_val *= tau_1 * (1 - attr_val) / click_prob;
            }

            click_probs[start_rank][ses_itr] = click_prob;
            // printf("%d, %d] click_probs[%d][%d] = %f\n", query_session.get_query(), tmp_sr.get_doc_id(), start_rank, ses_itr, click_prob);

            if (start_rank == 0) {
                exam_probs[ses_itr + 1] = exam_val;
            }

            ses_itr++;
        }
    }
}

DEV void CCM_Dev::compute_taus(int& thread_index, SERP& query_session, int& last_click_rank, float (&click_probs)[MAX_SERP_LENGTH][MAX_SERP_LENGTH], float (&exam_probs)[MAX_SERP_LENGTH + 1]) {
    float factor_values[8] = { 0.f };

    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++){
        SearchResult sr = query_session[rank];

        // double exam_probs_sum = 0.f;
        // double click_probs_sum = 0.f;
        // for (int i = 0; i < MAX_SERP_LENGTH; i++) {
        //     for (int j = 0; j < MAX_SERP_LENGTH - rank - 1; j++) {
        //         click_probs_sum += click_probs[i][j];
        //     }
        // }
        // for (int i = 0; i < MAX_SERP_LENGTH + 1; i++) {
        //     exam_probs_sum += exam_probs[i];
        // }
        // printf("%d, %d] factor init: click_probs sum = %f, exam_probs sum = %f, click = %d, last_click_rank = %d, rank = %d, attr = %f, tau 1 = %f, tau 2 = %f, tau 3 = %f\n", query_session.get_query(), sr.get_doc_id(), click_probs_sum, exam_probs_sum, sr.get_click(), last_click_rank, rank, this->attractiveness_parameters[sr.get_param_index()].value(), this->tau_parameters[0].value(), this->tau_parameters[1].value(), this->tau_parameters[2].value());

        // Send the initialization values to the phi function.
        CCMFactor factor_func(click_probs, exam_probs, sr.get_click(),
                              last_click_rank, rank,
                              this->attractiveness_parameters[sr.get_param_index()].value(),
                              this->tau_parameters[0].value(),
                              this->tau_parameters[1].value(),
                              this->tau_parameters[2].value());

        // // ! Current bug hypothesis
        // // ! All the previous results are correct, except that this version seems to take a line too many to be used for testing. This causes the seemingly missing lines in the result comparison.
        // // ! All input values for factor_func above seem to be correct, however according to the print statements below somehow the first couple of iterations are skipped with click_ and exam_probs. black magic
        // // ! Either the fault lies with wrong input values from click_ and/or exam_probs, or there are wrong calculations in factor.cu, because the input is correct (except for click_/exam_) but the
        // // ! new tau output isn't. These are the only parameters that differ.

        // printf("%d, %d] click = %d, last_click_rank = %d, rank = %d, attr = %f, tau 1 = %f, tau 2 = %f, tau 3 = %f\n", query_session.get_query(), sr.get_doc_id(), sr.get_click(), last_click_rank, rank, this->attractiveness_parameters[sr.get_param_index()].value(), this->tau_parameters[0].value(), this->tau_parameters[1].value(), this->tau_parameters[2].value());
        // for (int i = 0; i < MAX_SERP_LENGTH; i++) {
        //     for (int j = 0; j < MAX_SERP_LENGTH; j++) {
        //         printf("%d, %d] click_probs[%d][%d] = %f\n", query_session.get_query(), sr.get_doc_id(), i, j, click_probs[i][j]);
        //     }
        // }
        // for (int j = 0; j < MAX_SERP_LENGTH; j++) {
        //     printf("%d, %d] click_probs[%d][%d] = %f\n", query_session.get_query(), sr.get_doc_id(), 0, j, click_probs[0][j]);
        // }
        // if (query_session.get_query() == sr.get_doc_id()) {
        //     printf("found 0 0 !\n");
        //     for (int i = 0; i < MAX_SERP_LENGTH; i++) {
        //         for (int j = 0; j < MAX_SERP_LENGTH; j++) {
        //             printf("%d, %d] click_probs[%d][%d] = %f\n", query_session.get_query(), sr.get_doc_id(), i, j, click_probs[i][j]);
        //         }
        //     }
        // }
        // for (int j = 0; j < MAX_SERP_LENGTH + 1; j++) {
        //     printf("%d, %d] exam_probs[%d] = %f\n", query_session.get_query(), sr.get_doc_id(), j, exam_probs[j]);
        // }

        float factor_result = 0.f;
        float factor_sum = 0.f;

        // Compute phi for all possible input values.
        for (int fct_itr{0}; fct_itr < 8; fct_itr++) {
            factor_result = factor_func.compute(this->factor_inputs[fct_itr][0],
                                                this->factor_inputs[fct_itr][1],
                                                this->factor_inputs[fct_itr][2], query_session.get_query(), sr.get_doc_id());
                                                // this->factor_inputs[fct_itr][2]);
            factor_values[fct_itr] = factor_result;
            // printf("%d, %d] factor(%d, %d, %d) = %f\n", query_session.get_query(), sr.get_doc_id(), this->factor_inputs[fct_itr][0], this->factor_inputs[fct_itr][1], this->factor_inputs[fct_itr][2], factor_values[fct_itr]);
            factor_sum += factor_result;
        }


        if (sr.get_click() == 0) {
            this->compute_tau_1(thread_index, factor_values, factor_sum);

            // if (query_session.get_query() == 1421 && sr.get_doc_id() == 12596) {
            //     double numerator_update{(factor_values[5] + factor_values[7]) / factor_sum};
            //     double denominator_update{numerator_update + ((factor_values[4] + factor_values[6]) / factor_sum)};
            //     printf("%d, %d] new tau[%d] = %f / %f = %f (fv[5] = %f, fv[7] = %f, fv[4] = %f, fv[6] = %f, sum = %f)\n", query_session.get_query(), query_session[rank].get_doc_id(), 0, numerator_update, denominator_update, numerator_update/denominator_update, factor_values[5], factor_values[7], factor_values[4], factor_values[6], factor_sum);
            // }
            // printf("%d, %d] thread %d at index %d new tau[%d] = %f / %f = %f\n", query_session.get_query(), query_session[rank].get_doc_id(), thread_index, thread_index * 3 + 0, 0, numerator_update, denominator_update, numerator_update/denominator_update);

            // printf("%d, %d] new tau[%d] = ((%f + %f) / %f) / (%f + ((%f + %f) / %f)) = %f\n",
            //        query_session.get_query(), query_session[rank].get_doc_id(), 0,
            //        factor_values[5], factor_values[7], factor_sum, numerator_update, factor_values[4], factor_values[6], factor_sum,
            //        numerator_update/denominator_update);
        }
        else {
            this->compute_tau_2(thread_index, factor_values, factor_sum);

            // if (query_session.get_query() == 1421 && sr.get_doc_id() == 12596) {
            //     double numerator_update{factor_values[5] / factor_sum};
            //     double denominator_update{numerator_update + ((factor_values[4]) / factor_sum)};
            //     printf("%d, %d] new tau[%d] = %f / %f = %f (fv[5] = %f, fv[4] = %f, sum = %f)\n", query_session.get_query(), query_session[rank].get_doc_id(), 1, numerator_update, denominator_update, numerator_update/denominator_update, factor_values[5], factor_values[4], factor_sum);
            // }
            // printf("%d, %d] thread %d at index %d new tau[%d] = %f / %f = %f\n", query_session.get_query(), query_session[rank].get_doc_id(), thread_index, thread_index * 3 + 1, 1, numerator_update, denominator_update, numerator_update/denominator_update);

            this->compute_tau_3(thread_index, factor_values, factor_sum);

            // if (query_session.get_query() == 1421 && sr.get_doc_id() == 12596) {
            //     double numerator_update2{factor_values[7] / factor_sum};
            //     double denominator_update2{numerator_update2 + ((factor_values[6]) / factor_sum)};
            //     printf("%d, %d] new tau[%d] = %f / %f = %f (fv[7] = %f, fv[6] = %f, sum = %f)\n", query_session.get_query(), query_session[rank].get_doc_id(), 2, numerator_update2, denominator_update2, numerator_update2/denominator_update2, factor_values[7], factor_values[6], factor_sum);
            // }
            // printf("%d, %d] thread %d at index %d new tau[%d] = %f / %f = %f\n", query_session.get_query(), query_session[rank].get_doc_id(), thread_index, thread_index * 3 + 2, 2, numerator_update2, denominator_update2, numerator_update2/denominator_update2);
        }
    }
}

DEV void CCM_Dev::compute_tau_1(int& thread_index, float (&factor_values)[8], float& factor_sum) {
    double numerator_update{(factor_values[5] + factor_values[7]) / factor_sum};
    double denominator_update{numerator_update + ((factor_values[4] + factor_values[6]) / factor_sum)};
    this->tmp_tau_parameters[thread_index * 3 + 0].add_to_values(numerator_update, denominator_update);
    // this->tmp_tau_parameters[thread_index * 3 + 0].set_values(numerator_update, denominator_update);
}

DEV void CCM_Dev::compute_tau_2(int& thread_index, float (&factor_values)[8], float& factor_sum) {
    double numerator_update{factor_values[5] / factor_sum};
    double denominator_update{numerator_update + ((factor_values[4]) / factor_sum)};
    this->tmp_tau_parameters[thread_index * 3 + 1].add_to_values(numerator_update, denominator_update);
    // this->tmp_tau_parameters[thread_index * 3 + 1].set_values(numerator_update, denominator_update);
}

DEV void CCM_Dev::compute_tau_3(int& thread_index, float (&factor_values)[8], float& factor_sum) {
    double numerator_update{factor_values[7] / factor_sum};
    double denominator_update{numerator_update + ((factor_values[6]) / factor_sum)};
    this->tmp_tau_parameters[thread_index * 3 + 2].add_to_values(numerator_update, denominator_update);
    // this->tmp_tau_parameters[thread_index * 3 + 2].set_values(numerator_update, denominator_update);
}


/**
 * @brief Update the global parameter values using the local parameter values
 * on each thread.
 *
 * @param query_session The query session of this thread.
 * @param thread_index The index of the thread.
 * @param block_index The index of the block in which this thread exists.
 * @param parameter_type The type of parameter to update.
 * @param partition_size The size of the dataset.
 */
DEV void CCM_Dev::update_parameters(SERP& query_session, int& thread_index, int& block_index, int& parameter_type, int& partition_size) {
    this->update_tau_parameters(query_session, thread_index, block_index, partition_size);

    if (thread_index < partition_size) {
        this->update_attractiveness_parameters(query_session, thread_index);
    }
}

/**
 * @brief Update the global continuation parameters using the local continuation
 * parameters of a single thread.
 *
 * @param query_session The query session of this thread.
 * @param thread_index The index of this thread.
 * @param block_index The index of the block in which this thread exists.
 * @param partition_size The size of the dataset.
 */
DEV void CCM_Dev::update_tau_parameters(SERP& query_session, int& thread_index, int& block_index, int& partition_size) {
    // Initialize shared memory for this block's continuation parameters at 0.
    SHR float block_continuation_num[3];
    SHR float block_continuation_denom[3];
    // SHR float block_continuation_denom;
    // block_continuation_denom = 0.f;
    for (int tau_num = 0; tau_num < 3; tau_num++) {
        block_continuation_num[tau_num] = 0.f;
        block_continuation_denom[tau_num] = 0.f;
    }
    // Wait for all threads to finish initializing shared memory.
    __syncthreads();

    // Atomically add the values of the continuation parameters of this thread's
    // query session to the shared continuation parameters of this block.
    // Start every thread in this block at a different query session ranks
    // so prevent all threads from atomically writing to the same rank at the
    // same time.
    if (thread_index < partition_size) {
        int tau_num{0}, start_rank = block_index % 3;
        for (int offset = 0; offset < 3; offset++) {
            tau_num = (start_rank + offset) % 3;

            // Param tmp_tau = this->tmp_tau_parameters[thread_index * 3 + tau_num];
            // printf("%d] (thread = %d) Currently appending index %d to tau %d = %f / %f = %f\n", query_session.get_query(), thread_index, thread_index * 3 + tau_num, tau_num, tmp_tau.numerator_val(), tmp_tau.denominator_val(), tmp_tau.value());
            // if (!(tmp_tau.numerator_val() == 0.f && tmp_tau.denominator_val() == 0.f)) {
            atomicAddArch(&block_continuation_num[tau_num], this->tmp_tau_parameters[thread_index * 3 + tau_num].numerator_val());
            atomicAddArch(&block_continuation_denom[tau_num], this->tmp_tau_parameters[thread_index * 3 + tau_num].denominator_val());
            // }
            // atomicAddArch(&block_continuation_num[tau_num], this->tmp_tau_parameters[thread_index * 3 + tau_num].numerator_val());
            // atomicAddArch(&block_continuation_denom[tau_num], this->tmp_tau_parameters[thread_index * 3 + tau_num].denominator_val());

            // Atomically add the numerator and denominator values to shared memory.
            // atomicAddArch(&block_continuation_denom, 1.f / 3);
        }
    }
    // Wait for all threads to finish writing to shared memory.
    __syncthreads();
    // Have only the first few threads of the block write the shared memory
    // results to global memory.
    if (block_index < 3) {
        this->tau_parameters[block_index].atomic_add_to_values(block_continuation_num[block_index], block_continuation_denom[block_index]);
        // this->tau_parameters[block_index].add_to_values(block_continuation_num[block_index], block_continuation_denom);
    }

    // __syncthreads(); if (thread_index == 0) {
    //     printf("%d] new tau[0] = %f / %f = %f\n", thread_index, this->tau_parameters[0].numerator_val(), this->tau_parameters[0].denominator_val(), this->tau_parameters[0].value());
    //     printf("%d] new tau[1] = %f / %f = %f\n", thread_index, this->tau_parameters[1].numerator_val(), this->tau_parameters[1].denominator_val(), this->tau_parameters[1].value());
    //     printf("%d] new tau[2] = %f / %f = %f\n", thread_index, this->tau_parameters[2].numerator_val(), this->tau_parameters[2].denominator_val(), this->tau_parameters[2].value());
    // }
}

/**
 * @brief Update the global attractiveness parameters using the local
 * attractiveness parameters of a single thread.
 *
 * @param query_session The query session of this thread.
 * @param thread_index The index of this thread.
 */
DEV void CCM_Dev::update_attractiveness_parameters(SERP& query_session, int& thread_index) {
    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        SearchResult sr = query_session[rank];
        this->attractiveness_parameters[sr.get_param_index()].atomic_add_to_values(
            // this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].numerator_val(),
            // 1.f);
            this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].numerator_val(),
            this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].denominator_val());
        // printf("%d, %d] attr = %f / %f = %f\n", query_session.get_query(), sr.get_doc_id(), this->attractiveness_parameters[sr.get_param_index()].numerator_val(), this->attractiveness_parameters[sr.get_param_index()].denominator_val(), this->attractiveness_parameters[sr.get_param_index()].value());
        // printf("%d, %d] attr = %f / %f = %f\n", query_session.get_query(), sr.get_doc_id(),
        //     this->attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].numerator_val(),
        //     this->attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].denominator_val(),
        //     this->attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].value());
    }
}
