#include "hip/hip_runtime.h"
/** PBM click model.
 * Pooya Khandel's ParClick is used as a reference implementation.
 *
 * pbm.cu:
 *  - Defines the functions specific to creating a PBM CM.
 */

#include "pbm.cuh"


//---------------------------------------------------------------------------//
// Host-side PBM click model functions.                                      //
//---------------------------------------------------------------------------//

HST PBM_Hst::PBM_Hst() = default;

/**
 * @brief Constructs a PBM click model object for the host.
 *
 * @param pbm
 * @returns PBM_Hst The PBM click model object.
 */
HST PBM_Hst::PBM_Hst(PBM_Hst const &pbm) {
}

/**
 * @brief Creates a new PBM click model object.
 *
 * @return PBM_Hst* The PBM click model object.
 */
HST PBM_Hst* PBM_Hst::clone() {
    return new PBM_Hst(*this);
}

/**
 * @brief Print a message.
 */
HST void PBM_Hst::say_hello() {
    std::cout << "Host-side PBM says hello!" << std::endl;
}

/**
 * @brief Get the amount of device memory allocated to this click model.
 *
 * @return size_t The used memory.
 */
HST size_t PBM_Hst::get_memory_usage(void) {
    return this->cm_memory_usage;
}

/**
 * @brief Allocate device-side memory for the attractiveness parameters.
 *
 * @param partition The training and testing sets, and the number of
 * query-document pairs in the training set.
 * @param n_devices The number of devices on this node.
 */
HST void PBM_Hst::init_attractiveness_parameters(const std::tuple<std::vector<SERP>, std::vector<SERP>, int>& partition, int n_devices) {
    Param default_parameter;
    default_parameter.set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);

    // Allocate memory for the attractiveness parameters on the device.
    this->n_attr_dev = std::get<2>(partition);
    this->attractiveness_parameters.resize(this->n_attr_dev, default_parameter);
    CUDA_CHECK(hipMalloc(&this->attr_param_dptr, this->n_attr_dev * sizeof(Param)));
    CUDA_CHECK(hipMemcpy(this->attr_param_dptr, this->attractiveness_parameters.data(),
                          this->n_attr_dev * sizeof(Param), hipMemcpyHostToDevice));

    // Allocate memory for the temporary attractiveness parameters on the device.
    // These values are replaced at the start of each iteration, which means
    // they don't need to be initialized with a CUDA memory copy.
    this->n_tmp_attr_dev = std::get<0>(partition).size() * MAX_SERP_LENGTH;
    this->tmp_attractiveness_parameters.resize(this->n_tmp_attr_dev);
    CUDA_CHECK(hipMalloc(&this->tmp_attr_param_dptr, this->n_tmp_attr_dev * sizeof(Param)));

    // Store the number of allocated bytes.
    this->cm_memory_usage += this->n_attr_dev * sizeof(Param) + this->n_tmp_attr_dev * sizeof(Param);
}

/**
 * @brief Allocate device-side memory for the examination parameters.
 *
 * @param partition The training and testing sets, and the number of
 * query-document pairs in the training set.
 * @param n_devices The number of devices on this node.
 */
HST void PBM_Hst::init_examination_parameters(const std::tuple<std::vector<SERP>, std::vector<SERP>, int>& partition, int n_devices) {
    Param default_parameter;
    default_parameter.set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);

    // Allocate memory for the examination parameters on the device.
    this->n_exams_dev = MAX_SERP_LENGTH;
    this->examination_parameters.resize(this->n_exams_dev, default_parameter);
    CUDA_CHECK(hipMalloc(&this->exam_param_dptr, this->n_exams_dev * sizeof(Param)));
    CUDA_CHECK(hipMemcpy(this->exam_param_dptr, this->examination_parameters.data(),
                          this->n_exams_dev * sizeof(Param), hipMemcpyHostToDevice));

    // Allocate memory for the temporary examination parameters on the device.
    // These values are replaced at the start of each iteration, which means
    // they don't need to be initialized with a CUDA memory copy.
    this->n_tmp_exams_dev = std::get<0>(partition).size() * this->n_exams_dev;
    this->tmp_examination_parameters.resize(this->n_tmp_exams_dev);
    CUDA_CHECK(hipMalloc(&this->tmp_exam_param_dptr, this->n_tmp_exams_dev * sizeof(Param)));

    // Store the number of allocated bytes.
    this->cm_memory_usage += this->n_exams_dev * sizeof(Param) + this->n_tmp_exams_dev * sizeof(Param);
}

/**
 * @brief Allocate device-side memory for the attractiveness and examination
 * parameters of the click model.
 *
 * @param partition The training and testing sets, and the number of
 * query-document pairs in the training set.
 * @param n_devices The number of devices on this node.
 */
HST void PBM_Hst::init_parameters(const std::tuple<std::vector<SERP>, std::vector<SERP>, int>& partition, int n_devices) {
    this->init_attractiveness_parameters(partition, n_devices);
    this->init_examination_parameters(partition, n_devices);
}

/**
 * @brief Get the references to the allocated device-side memory.
 *
 * @param param_refs An array containing the references to the device-side
 * parameters in memory.
 * @param param_sizes The size of each of the memory allocations on the device.
 */
HST void PBM_Hst::get_device_references(Param**& param_refs, int*& param_sizes) {
    int n_references = 4;

    // Create a temporary array to store the device references.
    Param* tmp_param_refs_array[n_references];
    tmp_param_refs_array[0] = this->attr_param_dptr;
    tmp_param_refs_array[1] = this->tmp_attr_param_dptr;
    tmp_param_refs_array[2] = this->exam_param_dptr;
    tmp_param_refs_array[3] = this->tmp_exam_param_dptr;

    // Allocate space for the device references.
    CUDA_CHECK(hipMalloc(&param_refs, n_references * sizeof(Param*)));
    CUDA_CHECK(hipMemcpy(param_refs, tmp_param_refs_array,
                          n_references * sizeof(Param*), hipMemcpyHostToDevice));

    int tmp_param_sizes_array[n_references];
    tmp_param_sizes_array[0] = this->n_attr_dev;
    tmp_param_sizes_array[1] = this->n_tmp_attr_dev;
    tmp_param_sizes_array[2] = this->n_exams_dev;
    tmp_param_sizes_array[3] = this->n_tmp_exams_dev;

    // Allocate space for the device references.
    CUDA_CHECK(hipMalloc(&param_sizes, n_references * sizeof(int)));
    CUDA_CHECK(hipMemcpy(param_sizes, tmp_param_sizes_array,
                          n_references * sizeof(int), hipMemcpyHostToDevice));

    // Keep track of the pointers to the allocated device-side memory.
    this->param_refs = param_refs;
    this->param_sizes = param_sizes;
    this->cm_memory_usage += n_references * sizeof(Param*) + n_references * sizeof(int);
}

/**
 * @brief Update the global parameter values with the temporarily stored new
 * local parameter values on each thread.
 *
 * @param gridSize The size of kernel blocks on the GPU.
 * @param blockSize The number of kernel threads per block on the GPU.
 * @param partition The dataset allocated on the GPU.
 * @param dataset_size The size of the allocated dataset.
 */
HST void PBM_Hst::update_parameters(int& gridSize, int& blockSize, SERP*& partition, int& dataset_size) {
    Kernel::update<<<gridSize, blockSize>>>(partition, dataset_size);

    // for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
    //     typedef thrust::device_vector<int>::iterator Iterator;
    //     strided_range<Iterator> examination(this->tmp_exam_param_dptr + rank, this->tmp_exam_param_dptr + n_tmp_exams_dev, MAX_SERP_LENGTH);
    //     thrust::reduce(examination.begin(), examination.end())
    // }
}

/**
 * @brief Reset the original parameter values to zero so the previous parameter
 * values won't affect the next result twice.
 *
 * Further explanation; The first time it would affect the result would be when
 * retrieving their values in the training kernel, and the (unnecessary) second
 * time would be when adding the values to the original parameter containers.
 * The second time would still give a valid result but would slow down the
 * converging of the parameters.
 */
HST void PBM_Hst::reset_parameters(void) {
    // Create a parameter initialized at 0.
    Param default_parameter;
    default_parameter.set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);

    // Create an array of the right proportions with the empty parameters.
    std::vector<Param> cleared_examination_parameters(this->n_exams_dev, default_parameter);
    std::vector<Param> cleared_attractiveness_parameters(this->n_attr_dev, default_parameter);

    // Copy the cleared array to the device.
    CUDA_CHECK(hipMemcpy(this->exam_param_dptr, cleared_examination_parameters.data(), this->n_exams_dev * sizeof(Param), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(this->attr_param_dptr, cleared_attractiveness_parameters.data(), this->n_attr_dev * sizeof(Param), hipMemcpyHostToDevice));
}

/**
 * @brief Transfers parameters of a given type either from the device to the
 * host, or the otherway around.
 *
 * @param parameter_type The type of parameter that will be transfered.
 * (PUBLIC, PRIVATE, or ALL).
 * @param transfer_direction The direction in which the transfer will happen.
 * (H2D or D2H).
 */
HST void PBM_Hst::transfer_parameters(int parameter_type, int transfer_direction) {
    // Public parameters.
    if (parameter_type == PUBLIC || parameter_type == ALL) {
        if (transfer_direction == D2H) { // Transfer from device to host.
            // Retrieve the examination parameters from the device.
            CUDA_CHECK(hipMemcpy(this->examination_parameters.data(), this->exam_param_dptr, this->n_exams_dev * sizeof(Param), hipMemcpyDeviceToHost));
        }
        else if (transfer_direction == H2D) { // Transfer from host to device.
            // Send the examination parameters to the device.
            CUDA_CHECK(hipMemcpy(this->exam_param_dptr, this->examination_parameters.data(), this->n_exams_dev * sizeof(Param), hipMemcpyHostToDevice));
        }
    }

    // Private parameters.
    if (parameter_type == PRIVATE || parameter_type == ALL) {
        if (transfer_direction == D2H) { // Transfer from device to host.
            // Retrieve the attractiveness parameters from the device.
            CUDA_CHECK(hipMemcpy(this->attractiveness_parameters.data(), this->attr_param_dptr, this->n_attr_dev * sizeof(Param), hipMemcpyDeviceToHost));
        }
        else if (transfer_direction == H2D) { // Transfer from host to device.
            // Send the attractiveness parameters to the device.
            CUDA_CHECK(hipMemcpy(this->attr_param_dptr, this->attractiveness_parameters.data(), this->n_attr_dev * sizeof(Param), hipMemcpyHostToDevice));
        }
    }
}

/**
 * @brief Retrieve the parameters of a given type into a given array from the
 * click model.
 *
 * @param destination The array which will save the indicated parameters.
 * @param parameter_type The type of parameters which will be retrieved
 * (PUBLIC, PRIVATE, or ALL).
 */
HST void PBM_Hst::get_parameters(std::vector<std::vector<Param>>& destination, int parameter_type) {
    // Add the parameters to a generic vector which can represent  multiple
    // retrieved parameter types.
    if (parameter_type == PUBLIC) {
        destination.resize(1);
        destination[0] = this->examination_parameters;
    }
    else if (parameter_type == PRIVATE) {
        destination.resize(1);
        destination[0] = this->attractiveness_parameters;
    }
    else if (parameter_type == ALL) {
        destination.resize(2);
        destination[0] = this->examination_parameters;
        destination[1] = this->attractiveness_parameters;
    }
}

/**
 * @brief Compute the result of combining the PBM parameters from other nodes
 * or devices.
 *
 * @param parameters A multi-dimensional vector containing the parameters to be
 * combined. The vector is structured as follows: Node/Device ID -> Parameter
 * type -> Parameters.
 */
HST void PBM_Hst::sync_parameters(std::vector<std::vector<std::vector<Param>>>& parameters) {
    for (int rank = 0; rank < parameters[0][0].size(); rank++) {
        for (int param_type = 0; param_type < parameters[0].size(); param_type++) {
            Param base = parameters[0][param_type][rank];

            // Subtract the starting values of other partitions.
            parameters[0][param_type][rank].set_values(base.numerator_val() - (parameters.size() - 1),
                                                       base.denominator_val() - 2 * (parameters.size() - 1));

            for (int device_id = 1; device_id < parameters.size(); device_id++) {
                Param ex = parameters[device_id][param_type][rank];
                parameters[0][param_type][rank].add_to_values(ex.numerator_val(),
                                                              ex.denominator_val());
            }
        }
    }
}

/**
 * @brief Set the parameters of a host-side click model equal to the given
 * given arguments.
 *
 * @param source The new parameter values.
 * @param parameter_type The type of the given parameters. (PUBLIC, PRIVATE, or
 * ALL).
 */
HST void PBM_Hst::set_parameters(std::vector<std::vector<Param>>& source, int parameter_type) {
    // Set the parameters of this click model.
    if (parameter_type == PUBLIC) {
        this->examination_parameters = source[0];
    }
    else if (parameter_type == PRIVATE) {
        this->attractiveness_parameters = source[0];
    }
    else if (parameter_type == ALL) {
        this->examination_parameters = source[0];
        this->attractiveness_parameters = source[1];
    }
}

/**
 * @brief Compute the log-likelihood of the current PBM for the given query
 * session.
 *
 * @param query_session The query session for which the log-likelihood will be
 * computed.
 * @param log_click_probs The vector which will store the log-likelihood for
 * the document at each rank in the query session.
 */
HST void PBM_Hst::get_log_conditional_click_probs(SERP& query_session, std::vector<float>& log_click_probs) {
    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        SearchResult sr = query_session[rank];

        // Get the parameters corresponding to the current search result.
        // Return the default parameter value if the qd-pair was not found in
        // the training set.
        float atr{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        if (sr.get_param_index() != -1)
            atr = this->attractiveness_parameters[sr.get_param_index()].value();
        float ex{this->examination_parameters[sr.get_doc_rank()].value()};

        // Calculate the click probability.
        float atr_mul_ex = atr * ex;

        // Calculate the log click probability.
        int click{sr.get_click()};
        if (click == 1) {
            log_click_probs.push_back(std::log(atr_mul_ex));
        }
        else {
            log_click_probs.push_back(std::log(1 - atr_mul_ex));
        }
    }
}

/**
 * @brief Compute the click probability of the current PBM for the given query
 * session.
 *
 * @param query_session The query session for which the click probability will
 * be computed.
 * @param full_click_probs The vector which will store the click probability
 * for the document at each rank in the query session.
 */
HST void PBM_Hst::get_full_click_probs(SERP& query_session, std::vector<float> &full_click_probs) {
    // Go through all ranks of the query session.
    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        // Retrieve the search result at the current rank.
        SearchResult sr = query_session[rank];

        // Get the parameters corresponding to the current search result.
        // Return the default parameter value if the qd-pair was not found in
        // the training set.
        float atr{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        if (sr.get_param_index() != -1)
            atr = this->attractiveness_parameters[sr.get_param_index()].value();
        float ex{this->examination_parameters[sr.get_doc_rank()].value()};

        // Calculate the click probability.
        float atr_mul_ex = atr * ex;

        // Calculate the full click probability.
        int click{sr.get_click()};
        if (click == 1) {
            full_click_probs.push_back(atr_mul_ex);
        }
        else {
            full_click_probs.push_back(1 - atr_mul_ex);
        }
    }
}

/**
 * @brief Frees the memory allocated to the parameters of this click model on
 * the GPU device.
 */
HST void PBM_Hst::destroy_parameters(void) {
    // Free origin and temporary attractiveness containers.
    CUDA_CHECK(hipFree(this->attr_param_dptr));
    CUDA_CHECK(hipFree(this->tmp_attr_param_dptr));

    // Free origin and temporary examination containers.
    CUDA_CHECK(hipFree(this->exam_param_dptr));
    CUDA_CHECK(hipFree(this->tmp_exam_param_dptr));

    // Free the device parameter references and sizes.
    CUDA_CHECK(hipFree(this->param_refs));
    CUDA_CHECK(hipFree(this->param_sizes));

    // Reset used device memory.
    this->cm_memory_usage = 0;
}


//---------------------------------------------------------------------------//
// Device-side PBM click model functions.                                    //
//---------------------------------------------------------------------------//

/**
 * @brief Prints a message.
 */
DEV void PBM_Dev::say_hello() {
    printf("Device-side PBM says hello!\n");
}

/**
 * @brief Creates a new PBM click model object.
 *
 * @return PBM_Dev* The PBM click model object.
 */
DEV PBM_Dev *PBM_Dev::clone() {
    return new PBM_Dev(*this);
}

DEV PBM_Dev::PBM_Dev() = default;

/**
 * @brief Constructs a PBM click model object for the device.
 *
 * @param pbm
 * @returns PBM_Dev The PBM click model object.
 */
DEV PBM_Dev::PBM_Dev(PBM_Dev const &pbm) {
}

/**
 * @brief Set the location of the memory allocated for the parameters of this
 * click model on the GPU device.
 *
 * @param parameter_ptr The pointers to the allocated memory.
 * @param parameter_sizes The size of the allocated memory.
 */
DEV void PBM_Dev::set_parameters(Param**& parameter_ptr, int* parameter_sizes) {
    this->attractiveness_parameters = parameter_ptr[0];
    this->tmp_attractiveness_parameters = parameter_ptr[1];
    this->examination_parameters = parameter_ptr[2];
    this->tmp_examination_parameters = parameter_ptr[3];

    this->n_attractiveness_parameters = parameter_sizes[0];
    this->n_tmp_attractiveness_parameters = parameter_sizes[1];
    this->n_examination_parameters = parameter_sizes[2];
    this->n_tmp_examination_parameters = parameter_sizes[3];
}

/**
 * @brief Compute a single Expectation-Maximization iteration for the PBM click
 * model, for a single query session.
 *
 * @param query_session The query session which will be used to estimate the
 * PBM parameters.
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 */
DEV void PBM_Dev::process_session(SERP& query_session, int& thread_index) {
    // int query_id = query_session.get_query();
    // printf("%d\n", query_session.get_query());
    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        SearchResult sr = query_session[rank];

        // Get the attractiveness and examination parameters.
        float atr{this->attractiveness_parameters[sr.get_param_index()].value()};
        float ex{this->examination_parameters[rank].value()};

        // Set the default values of the attractiveness and examination
        // parameters. These will be the parameter values in case the search
        // result document has been clicked.
        float new_numerator_atr{1};
        float new_numerator_ex{1};

        // If the search result document hasn't been clicked, then calculate
        // estimate the parameter value.
        if (sr.get_click() == 0) {
            // Calculate the current qd-pair click probability.
            float atr_ex = atr * ex;

            // Attractiveness = ((1 - gamma_{r}^{(t)}) * alpha_{qd}^{(t)}) / (1 - (gamma_{r}^{(t)} * alpha_{qd}^{(t)}))
            new_numerator_atr = (atr - atr_ex) / (1 - atr_ex);
            // Examination = ((1 - alpha_{qd}^{(t)}) * gamma_{r}^{(t)}) / (1 - (gamma_{r}^{(t)} * alpha_{qd}^{(t)}))
            new_numerator_ex = (ex - atr_ex) / (1 - atr_ex);
        }

        // Store the temporary attractiveness and examination parameters.
        this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].set_values(new_numerator_atr, 1);
        this->tmp_examination_parameters[thread_index * MAX_SERP_LENGTH + rank].set_values(new_numerator_ex, 1);
    }
}

/**
 * @brief Update the global parameter values using the local parameter values
 * on each thread.
 *
 * @param query_session The query session of this thread.
 * @param thread_index The index of the thread.
 * @param block_index The index of the block in which this thread exists.
 * @param parameter_type The type of parameter to update.
 * @param partition_size The size of the dataset.
 */
DEV void PBM_Dev::update_parameters(SERP& query_session, int& thread_index, int& block_index, int& partition_size) {
    this->update_examination_parameters(query_session, thread_index, block_index, partition_size);

    if (thread_index < partition_size) {
        this->update_attractiveness_parameters(query_session, thread_index);
    }
}

/**
 * @brief Update the global examination parameters using the local examination
 * parameters of a single thread.
 *
 * @param query_session The query session of this thread.
 * @param thread_index The index of this thread.
 * @param block_index The index of the block in which this thread exists.
 * @param partition_size The size of the dataset.
 */
DEV void PBM_Dev::update_examination_parameters(SERP& query_session, int& thread_index, int& block_index, int& partition_size) {
    // Initialize shared memory for this block's examination parameters at 0.
    SHR float block_examination_num[MAX_SERP_LENGTH];
    SHR float block_examination_denom;
    block_examination_denom = 0.f;
    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        block_examination_num[rank] = 0.f;
    }
    // Wait for all threads to finish initializing shared memory.
    __syncthreads();

    // Atomically add the values of the examination parameters of this thread's
    // query session to the shared examination parameters of this block.
    // Start every thread in this block at a different query session ranks
    // so prevent all threads from atomically writing to the same rank at the
    // same time.
    if (thread_index < partition_size) {
        int rank{0}, start_rank = block_index % MAX_SERP_LENGTH;
        for (int offset = 0; offset < MAX_SERP_LENGTH; offset++) {
            rank = (start_rank + offset) % MAX_SERP_LENGTH;

            // Atomically add the numerator and denominator values to shared memory.
            atomicAddArch(&block_examination_num[rank], this->tmp_examination_parameters[thread_index * MAX_SERP_LENGTH + rank].numerator_val());
            atomicAddArch(&block_examination_denom, 1.f / MAX_SERP_LENGTH);
        }
    }
    // Wait for all threads to finish writing to shared memory.
    __syncthreads();
    // Have only the first few threads of the block write the shared memory
    // results to global memory.
    if (block_index < MAX_SERP_LENGTH) {
        this->examination_parameters[block_index].atomic_add_to_values(block_examination_num[block_index], block_examination_denom);
    }
}

/**
 * @brief Update the global attractiveness parameters using the local
 * attractiveness parameters of a single thread.
 *
 * @param query_session The query session of this thread.
 * @param thread_index The index of this thread.
 */
DEV void PBM_Dev::update_attractiveness_parameters(SERP& query_session, int& thread_index) {
    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        SearchResult sr = query_session[rank];
        this->attractiveness_parameters[sr.get_param_index()].atomic_add_to_values(
            this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].numerator_val(),
            1.f);
    }
}
