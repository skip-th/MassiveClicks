#include "hip/hip_runtime.h"
/** DBN click model.
 *
 * dbn.cu:
 *  - Defines the functions specific to creating a DBN CM.
 */

#include "dbn.cuh"


//---------------------------------------------------------------------------//
// Host-side DBN click model functions.                                      //
//---------------------------------------------------------------------------//

HST DBN_Hst::DBN_Hst() = default;

/**
 * @brief Constructs a DBN click model object for the host.
 *
 * @param dbn The base click model object to copy.
 * @return The DBN click model object.
 */
HST DBN_Hst::DBN_Hst(DBN_Hst const &dbn) {
}

/**
 * @brief Creates a new DBN click model object.
 *
 * @return The DBN click model object.
 */
HST DBN_Hst* DBN_Hst::clone() {
    return new DBN_Hst(*this);
}

/**
 * @brief Print a message.
 */
HST void DBN_Hst::say_hello() {
    std::cout << "Host-side DBN says hello!" << std::endl;
}

/**
 * @brief Get the amount of device memory allocated to this click model.
 *
 * @return The used memory.
 */
HST size_t DBN_Hst::get_memory_usage(void) {
    return this->cm_memory_usage;
}

/**
 * @brief Get the expected amount of memory the click model will need to store
 * the current parameters.
 *
 * @param n_queries The number of queries assigned to this click model.
 * @param n_qd The number of query-document pairs assigned to this click model.
 * @return The worst-case parameter memory footprint.
 */
HST size_t DBN_Hst::compute_memory_footprint(int n_queries, int n_qd) {
    std::pair<int, int> n_attractiveness = this->get_n_atr_params(n_queries, n_qd);
    std::pair<int, int> n_satisfaction = this->get_n_sat_params(n_queries, n_qd);
    std::pair<int, int> n_continuation = this->get_n_gam_params(n_queries, n_qd);

    return (n_attractiveness.first + n_attractiveness.second +
            n_satisfaction.first + n_satisfaction.second +
            n_continuation.first + n_continuation.second) * sizeof(Param);
}

/**
 * @brief Get the number of original and temporary attractiveness parameters.
 *
 * @param n_queries The number of queries assigned to this click model.
 * @param n_qd The number of query-document pairs assigned to this click model.
 * @return The number of original and temporary examination
 * parameters.
 */
HST std::pair<int,int> DBN_Hst::get_n_atr_params(int n_queries, int n_qd) {
    return std::make_pair(n_qd,                  // # original
                          n_queries * MAX_SERP); // # temporary
}

/**
 * @brief Get the number of original and temporary satisfaction parameters.
 *
 * @param n_queries The number of queries assigned to this click model.
 * @param n_qd The number of query-document pairs assigned to this click model.
 * @return The number of original and temporary satisfaction
 * parameters.
 */
HST std::pair<int, int> DBN_Hst::get_n_sat_params(int n_queries, int n_qd) {
    return std::make_pair(n_qd,                  // # original
                          n_queries * MAX_SERP); // # temporary
}

/**
 * @brief Get the number of original and temporary continuation (gamma) parameters.
 *
 * @param n_queries The number of queries assigned to this click model.
 * @param n_qd The number of query-document pairs assigned to this click model.
 * @return The number of original and temporary continuation
 * parameters.
 */
HST std::pair<int, int> DBN_Hst::get_n_gam_params(int n_queries, int n_qd) {
    return std::make_pair(N_GAM,              // # original
                          n_queries * N_GAM); // # temporary
}

/**
 * @brief Allocate device-side memory for the attractiveness, satisfaction and
 * continuation parameters of the click model.
 *
 * @param dataset The training and testing sets, and the number of
 * query-document pairs in the training set.
 * @param n_devices The number of devices on this node.
 * @param fmem The amount of free memory on the device.
 * @param device The device to allocate memory on.
 */
HST void DBN_Hst::init_parameters(const Partition& dataset, const size_t fmem, const bool device) {
    std::pair<int, int> n_attractiveness = this->get_n_atr_params(std::get<0>(dataset).size(), std::get<2>(dataset));
    init_parameters_hst(this->atr_parameters, this->atr_tmp_parameters, this->atr_dptr, this->atr_tmp_dptr, n_attractiveness, this->n_atr_params, this->n_atr_tmp_params, this->cm_memory_usage, dataset, fmem, device);
    std::pair<int, int> n_satisfaction = this->get_n_sat_params(std::get<0>(dataset).size(), std::get<2>(dataset));
    init_parameters_hst(this->sat_parameters, this->sat_tmp_parameters, this->sat_dptr, this->sat_tmp_dptr, n_satisfaction, this->n_sat_params, this->n_sat_tmp_params, this->cm_memory_usage, dataset, fmem, device);
    std::pair<int, int> n_continuation = this->get_n_gam_params(std::get<0>(dataset).size(), std::get<2>(dataset));
    init_parameters_hst(this->gam_parameters, this->gam_tmp_parameters, this->gam_dptr, this->gam_tmp_dptr, n_continuation, this->n_gam_params, this->n_gam_tmp_params, this->cm_memory_usage, dataset, fmem, device);
}

/**
 * @brief Get the name of the parameters of this click model.
 *
 * @return The public and private parameter names.
 */
HST void DBN_Hst::get_parameter_information(
        std::pair<std::vector<std::string>, std::vector<std::string>> &headers,
        std::pair<std::vector<std::vector<Param> *>, std::vector<std::vector<Param> *>> &parameters) {
    // Set parameter headers.
    std::vector<std::string> public_name = {"continuation"};
    std::vector<std::string> private_name = {"attractiveness", "satisfaction"};
    headers = std::make_pair(public_name, private_name);

    // Set parameter values.
    std::vector<std::vector<Param> *> public_parameters = {&this->gam_parameters};
    std::vector<std::vector<Param> *> private_parameters = {&this->atr_parameters, &this->sat_parameters};
    parameters = std::make_pair(public_parameters, private_parameters);
}

/**
 * @brief Get the references to the allocated device-side memory.
 *
 * @param param_refs An array containing the references to the device-side
 * parameters in memory.
 * @param param_sizes The size of each of the memory allocations on the device.
 */
HST void DBN_Hst::get_device_references(Param**& param_refs, int*& param_sizes) {
    int n_references = 6;

    // Create a temporary array to store the device references.
    Param* tmp_param_refs_array[n_references];
    tmp_param_refs_array[0] = this->atr_dptr;
    tmp_param_refs_array[1] = this->atr_tmp_dptr;
    tmp_param_refs_array[2] = this->sat_dptr;
    tmp_param_refs_array[3] = this->sat_tmp_dptr;
    tmp_param_refs_array[4] = this->gam_dptr;
    tmp_param_refs_array[5] = this->gam_tmp_dptr;

    // Allocate space for the device references.
    CUDA_CHECK(hipMalloc(&param_refs, n_references * sizeof(Param*)));
    CUDA_CHECK(hipMemcpy(param_refs, tmp_param_refs_array,
                          n_references * sizeof(Param*), hipMemcpyHostToDevice));

    int tmp_param_sizes_array[n_references];
    tmp_param_sizes_array[0] = this->n_atr_params;
    tmp_param_sizes_array[1] = this->n_atr_tmp_params;
    tmp_param_sizes_array[2] = this->n_sat_params;
    tmp_param_sizes_array[3] = this->n_sat_tmp_params;
    tmp_param_sizes_array[4] = this->n_gam_params;
    tmp_param_sizes_array[5] = this->n_gam_tmp_params;

    // Allocate space for the device references.
    CUDA_CHECK(hipMalloc(&param_sizes, n_references * sizeof(int)));
    CUDA_CHECK(hipMemcpy(param_sizes, tmp_param_sizes_array,
                          n_references * sizeof(int), hipMemcpyHostToDevice));

    // Keep track of the pointers to the allocated device-side memory.
    this->param_refs = param_refs;
    this->param_sizes = param_sizes;
    this->cm_memory_usage += n_references * sizeof(Param*) + n_references * sizeof(int);
}

/**
 * @brief Update the global parameter values using the temporary parameters.
 *
 * @param dataset The training set.
 * @param thread_start_idx Dataset starting indices of each thread.
 */
HST void DBN_Hst::update_parameters(TrainSet& dataset, const std::vector<int>& thread_start_idx) {
    update_unique_parameters_hst(this->atr_tmp_parameters, this->atr_parameters, dataset, thread_start_idx);
    update_unique_parameters_hst(this->sat_tmp_parameters, this->sat_parameters, dataset, thread_start_idx);
    update_shared_parameters_hst(this->gam_tmp_parameters, this->gam_parameters, dataset, thread_start_idx);
}

/**
 * @brief Compute a single Expectation-Maximization iteration for the DBN click
 * model for each query session.
 *
 * @param dataset The training set.
 * @param thread_start_idx Dataset starting indices of each thread.
 */
HST void DBN_Hst::process_session(const TrainSet& dataset, const std::vector<int>& thread_start_idx) {
    // Iterate over the queries in the dataset in each thread.
    auto process_session_thread = [this](const TrainSet& dataset, const int thread_idx, int start_idx, int stop_idx) {
        int dataset_size = dataset.size();

        for (int query_index = start_idx; query_index < stop_idx; query_index++) {
            // Retrieve the search results associated with the current query.
            SERP_Hst query_session = dataset[query_index];

            int last_click_rank = query_session.last_click_rank();
            float click_probs[MAX_SERP][MAX_SERP] = { 0.f };
            float exam_probs[MAX_SERP + 1];
            float exam[MAX_SERP + 1];
            float car[MAX_SERP + 1] = { 0.f };

            this->gam_tmp_parameters[query_index].set_values(0.f, 0.f);

            this->compute_exm_car(query_session, exam, car);
            this->compute_dbn_atr(query_index, query_session, last_click_rank, exam, car, dataset_size);
            this->compute_dbn_sat(query_index, query_session, last_click_rank, car, dataset_size);
            this->get_tail_clicks(query_index, query_session, click_probs, exam_probs);
            this->compute_gamma(query_index, query_session, last_click_rank, click_probs, exam_probs);
        }
    };

    // Create threads.
    int n_threads = thread_start_idx.size();
    std::thread threads[n_threads];

    // Divide queries among threads.
    int thread_part = dataset.size() / n_threads;
    int thread_part_left = dataset.size() % n_threads;
    int start_idx{0}, stop_idx{0};

    // Launch threads.
    for (int tid = 0; tid < n_threads; tid++) {
        stop_idx += tid < thread_part_left ? thread_part + 1 : thread_part;
        threads[tid] = std::thread(process_session_thread, dataset, tid, start_idx, stop_idx);
        start_idx += tid < thread_part_left ? thread_part + 1 : thread_part;
    }

    // Join threads.
    for (int tid = 0; tid < n_threads; tid++) {
        threads[tid].join();
    }
}

/**
 * @brief Compute the examination parameter for every rank of this query
 * session. The examination parameter can be re-computed every iteration using
 * the values from attractiveness, satisfaction, and continuation parameters
 * from the previous iteration.
 *
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param exam The examination parameters for every rank. The first rank is
 * always examined (1).
 * @param car
 */
HST void DBN_Hst::compute_exm_car(SERP_Hst& query_session, float (&exam)[MAX_SERP + 1], float (&car)[MAX_SERP + 1]) {
    // Set the default examination value for the first rank.
    exam[0] = 1.f;

    float attr_val, sat_value, gamma_value, ex_value, temp, car_val;
    float car_helper[MAX_SERP][2];

    for (int rank = 0; rank < MAX_SERP;) {
        SearchResult_Hst sr = query_session[rank];

        attr_val = this->atr_parameters[sr.get_param_index()].value();
        sat_value = this->sat_parameters[sr.get_param_index()].value();
        gamma_value = this->gam_parameters[0].value();
        ex_value = exam[rank];

        temp = gamma_value * (1 - attr_val);
        ex_value *= temp + gamma_value * attr_val * (1 - sat_value);

        car_helper[rank][0] = attr_val;
        car_helper[rank][1] = temp;

        rank += 1;
        exam[rank] = ex_value;
    }

    for (int car_itr = MAX_SERP - 1; car_itr > -1; car_itr--) {
        car_val = car[car_itr + 1];

        car[car_itr] = car_helper[car_itr][0] + car_helper[car_itr][1] * car_val;
    }
}

/**
 * @brief Compute the attractiveness parameter for every rank of this query
 * session.
 *
 * @param qid The index of the query session in the dataset.
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param last_click_rank The last rank of this query sessions which has been
 * clicked.
 * @param exam The examination parameters for every rank. The first rank is
 * always examined (1).
 * @param car
 * @param dataset_size The size of the dataset.
 */
HST void DBN_Hst::compute_dbn_atr(int& qid, SERP_Hst& query_session, int& last_click_rank, float (&exam)[MAX_SERP + 1], float (&car)[MAX_SERP + 1], int& dataset_size) {
    float numerator_update, denominator_update;
    float exam_val, attr_val,  car_val;

    #pragma unroll
    for (int rank = 0; rank < MAX_SERP; rank++) {
        SearchResult_Hst sr = query_session[rank];

        numerator_update = 0.f;
        denominator_update = 1.f;

        if (sr.get_click() == 1) {
            numerator_update += 1.f;
        }
        else if (rank >= last_click_rank) {
            attr_val = this->atr_parameters[sr.get_param_index()].value();
            exam_val = exam[rank];
            car_val = car[rank];

            numerator_update += (attr_val * (1 - exam_val)) / (1 - exam_val * car_val);
        }

        this->atr_tmp_parameters[rank * dataset_size + qid].set_values(numerator_update, denominator_update);
    }
}

/**
 * @brief Compute the satisfaction parameter for every rank of this query
 * session.
 *
 * @param qid The index of the query session in the dataset.
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param last_click_rank The last rank of this query sessions which has been
 * clicked.
 * @param car
 * @param dataset_size The size of the dataset.
 */
HST void DBN_Hst::compute_dbn_sat(int& qid, SERP_Hst& query_session, int& last_click_rank, float (&car)[MAX_SERP + 1], int& dataset_size) {
    float numerator_update, denominator_update;
    float gamma_val, sat_val, car_val;

    #pragma unroll
    for (int rank = 0; rank < MAX_SERP; rank++) {
        SearchResult_Hst sr = query_session[rank];

        if (sr.get_click() == 1) {
            numerator_update = 0.f;
            denominator_update = 1.f;

            if (rank == last_click_rank) {
                sat_val = this->sat_parameters[sr.get_param_index()].value();
                gamma_val = this->gam_parameters[0].value();

                if (rank < MAX_SERP - 1) {
                    car_val = car[rank + 1];
                } else{
                    car_val = 0.f;
                }

                numerator_update += sat_val / (1 - (1 - sat_val) * gamma_val * car_val);
            }

            this->sat_tmp_parameters[rank * dataset_size + qid].set_values(numerator_update, denominator_update);
        }
    }
}

/**
 * @brief Compute the click probabilities of a rank given the clicks on the
 * preceding ranks.
 *
 * @param qid The index of the query session in the dataset.
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param click_probs The probabilty of a click occurring on a rank.
 * @param exam_probs The probability of a rank being examined.
 */
HST void DBN_Hst::get_tail_clicks(int& qid, SERP_Hst& query_session, float (&click_probs)[MAX_SERP][MAX_SERP], float (&exam_probs)[MAX_SERP + 1]) {
    exam_probs[0] = 1.f;
    float exam_val, gamma_val, click_prob;

    for (int start_rank = 0; start_rank < MAX_SERP; start_rank++) {
        exam_val = 1.f;

        int ses_itr{0};
        for (int res_itr = start_rank; res_itr < MAX_SERP; res_itr++) {
            SearchResult_Hst tmp_sr = query_session[ses_itr];

            float attr_val = this->atr_parameters[tmp_sr.get_param_index()].value();
            float sat_val = this->sat_parameters[tmp_sr.get_param_index()].value();
            gamma_val = this->gam_parameters[0].value();

            if (query_session[res_itr].get_click() == 1) {
                click_prob = attr_val * exam_val;
                exam_val = gamma_val * (1 - sat_val);
            }
            else{
                click_prob = 1 - attr_val * exam_val;
                exam_val *= gamma_val * (1 - attr_val) / click_prob;
            }

            click_probs[start_rank][ses_itr] = click_prob;

            if (start_rank == 0) {
                exam_probs[ses_itr + 1] = exam_val;
            }

            ses_itr++;
        }
    }
}

/**
 * @brief Compute the continuation parameter gamma.
 *
 * @param qid The index of the query session in the dataset.
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param last_click_rank The last rank of this query sessions which has been
 * clicked.
 * @param click_probs The probabilty of a click occurring on a rank.
 * @param exam_probs The probability of a rank being examined.
 */
HST void DBN_Hst::compute_gamma(int& qid, SERP_Hst& query_session, int& last_click_rank, float (&click_probs)[MAX_SERP][MAX_SERP], float (&exam_probs)[MAX_SERP + 1]) {
    float factor_values[8] = { 0.f };

    #pragma unroll
    for (int rank = 0; rank < MAX_SERP; rank++){
        SearchResult_Hst sr = query_session[rank];

        // Send the initialization values to the phi function.
        DBNFactor factor_func(click_probs, exam_probs, sr.get_click(),
                              last_click_rank, rank,
                              this->atr_parameters[sr.get_param_index()].value(),
                              this->sat_parameters[sr.get_param_index()].value(),
                              this->gam_parameters[0].value());

        float factor_result = 0.f;
        float factor_sum = 0.f;

        // Compute phi for all possible input values.
        for (int fct_itr{0}; fct_itr < 8; fct_itr++) {
            factor_result = factor_func.compute(this->factor_inputs[fct_itr][0],
                                                this->factor_inputs[fct_itr][1],
                                                this->factor_inputs[fct_itr][2]);
            factor_values[fct_itr] = factor_result;
            factor_sum += factor_result;
        }

        float numerator_update = factor_values[5] / factor_sum;
        float denominator_update = (factor_values[4] + factor_values[5]) / factor_sum;

        this->gam_tmp_parameters[qid].add_to_values(numerator_update, denominator_update);
    }
}

/**
 * @brief Reset the original parameter values to zero so the previous parameter
 * values won't affect the next result twice.
 *
 * Further explanation; The first time it would affect the result would be when
 * retrieving their values in the training kernel, and the (unnecessary) second
 * time would be when adding the values to the original parameter containers.
 * The second time would still give a valid result but would slow down the
 * converging of the parameters.
 *
 * @param device Whether to reset the device parameters or the host parameters.
 * (true for device, false for host).
 */
HST void DBN_Hst::reset_parameters(bool device) {
    reset_parameters_hst(this->sat_parameters, this->sat_dptr, device);
    reset_parameters_hst(this->atr_parameters, this->atr_dptr, device);
    reset_parameters_hst(this->gam_parameters, this->gam_dptr, device);
}

/**
 * @brief Transfers parameters of a given type either from the device to the
 * host, or the otherway around.
 *
 * @param parameter_type The type of parameter that will be transfered.
 * (PUBLIC, PRIVATE, or ALL).
 * @param transfer_direction The direction in which the transfer will happen.
 * (H2D or D2H).
 * @param tmp Whether to transfer the temporary parameters or the originals.
 */
HST void DBN_Hst::transfer_parameters(int parameter_type, int transfer_direction, bool tmp) {
    // Public parameters.
    if (parameter_type == PUBLIC || parameter_type == ALL) {
        if (tmp) transfer_parameters_hst(transfer_direction, this->gam_tmp_parameters, this->gam_tmp_dptr);
        if (!tmp) transfer_parameters_hst(transfer_direction, this->gam_parameters, this->gam_dptr);
    }

    // Private parameters.
    if (parameter_type == PRIVATE || parameter_type == ALL) {
        if (tmp) transfer_parameters_hst(transfer_direction, this->sat_tmp_parameters, this->sat_tmp_dptr);
        if (!tmp) transfer_parameters_hst(transfer_direction, this->sat_parameters, this->sat_dptr);
        if (tmp) transfer_parameters_hst(transfer_direction, this->atr_tmp_parameters, this->atr_tmp_dptr);
        if (!tmp) transfer_parameters_hst(transfer_direction, this->atr_parameters, this->atr_dptr);
    }
}

/**
 * @brief Retrieve the parameters of a given type into a given array from the
 * click model.
 *
 * @param destination The array which will save the indicated parameters.
 * @param parameter_type The type of parameters which will be retrieved
 * (PUBLIC, PRIVATE, or ALL).
 */
HST void DBN_Hst::get_parameters(std::vector<std::vector<Param>>& destination, int parameter_type) {
    // Add the parameters to a generic vector which can represent  multiple
    // retrieved parameter types.
    if (parameter_type == PUBLIC) {
        destination.resize(1);
        destination[0] = this->gam_parameters;
    }
    else if (parameter_type == PRIVATE) {
        destination.resize(2);
        destination[0] = this->atr_parameters;
        destination[1] = this->sat_parameters;
    }
    else if (parameter_type == ALL) {
        destination.resize(3);
        destination[0] = this->atr_parameters;
        destination[1] = this->sat_parameters;
        destination[2] = this->gam_parameters;
    }
}

/**
 * @brief Set the parameters of a host-side click model equal to the given
 * given arguments.
 *
 * @param source The new parameter values.
 * @param parameter_type The type of the given parameters. (PUBLIC, PRIVATE, or
 * ALL).
 */
HST void DBN_Hst::set_parameters(std::vector<std::vector<Param>>& source, int parameter_type) {
    // Set the parameters of this click model.
    if (parameter_type == PUBLIC) {
        this->gam_parameters = source[0];
    }
    else if (parameter_type == PRIVATE) {
        this->atr_parameters = source[0];
        this->sat_parameters = source[1];
    }
    else if (parameter_type == ALL) {
        this->atr_parameters = source[0];
        this->sat_parameters = source[1];
        this->gam_parameters = source[2];
    }
}

/**
 * @brief Get probability of a click on a search result.
 *
 * @param query_session The query session containing the search results.
 * @param probabilities The probabilities of a click on each search result.
 */
HST void DBN_Hst::get_serp_probability(SERP_Hst& query_session, float (&probablities)[MAX_SERP]) {
    float ex{1.f}, click_prob;

    #pragma unroll
    for (int rank = 0; rank < MAX_SERP; rank++) {
        SearchResult_Hst sr = query_session[rank];

        // Get the parameters corresponding to the current search result.
        // Return the default parameter value if the qd-pair was not found in
        // the training set.
        float attr_val{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        float sat_val{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        if (sr.get_param_index() != -1) {
            attr_val = this->atr_parameters[sr.get_param_index()].value();
            sat_val = this->sat_parameters[sr.get_param_index()].value();
        }
        float gamma_val{this->gam_parameters[0].value()};

        if (sr.get_click() == 1) {
            click_prob = attr_val * ex;
            ex = gamma_val * ( 1- sat_val);
        } else{
            click_prob = 1 - attr_val * ex;
            ex *= gamma_val * ( 1 - attr_val) / click_prob;
        }
        // Calculate the click probability.
        probablities[rank] = click_prob;
    }
}

/**
 * @brief Compute the log-likelihood of the current DBN for the given query
 * session.
 *
 * @param query_session The query session for which the log-likelihood will be
 * computed.
 * @param log_click_probs The vector which will store the log-likelihood for
 * the document at each rank in the query session.
 */
HST void DBN_Hst::get_log_conditional_click_probs(SERP_Hst& query_session, std::vector<float>& log_click_probs) {
    float ex{1.f}, click_prob;

    #pragma unroll
    for (int rank = 0; rank < MAX_SERP; rank++) {
        SearchResult_Hst sr = query_session[rank];

        // Get the parameters corresponding to the current search result.
        // Return the default parameter value if the qd-pair was not found in
        // the training set.
        float attr_val{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        float sat_val{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        if (sr.get_param_index() != -1) {
            attr_val = this->atr_parameters[sr.get_param_index()].value();
            sat_val = this->sat_parameters[sr.get_param_index()].value();
        }
        float gamma_val{this->gam_parameters[0].value()};

        if (sr.get_click() == 1) {
            click_prob = attr_val * ex;
            ex = gamma_val * ( 1- sat_val);
        } else{
            click_prob = 1 - attr_val * ex;
            ex *= gamma_val * ( 1 - attr_val) / click_prob;
        }

        log_click_probs.push_back(std::log(click_prob));
    }
}

/**
 * @brief Compute the click probability of the current DBN for the given query
 * session.
 *
 * @param query_session The query session for which the click probability will
 * be computed.
 * @param full_click_probs The vector which will store the click probability
 * for the document at each rank in the query session.
 */
HST void DBN_Hst::get_full_click_probs(SERP_Hst& query_session, std::vector<float> &full_click_probs) {
    float ex{1.f}, atr_mul_ex;

    // Go through all ranks of the query session.
    #pragma unroll
    for (int rank = 0; rank < MAX_SERP; rank++) {
        // Retrieve the search result at the current rank.
        SearchResult_Hst sr = query_session[rank];

        // Get the parameters corresponding to the current search result.
        // Return the default parameter value if the qd-pair was not found in
        // the training set.
        float atr{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        float sat{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        if (sr.get_param_index() != -1) {
            atr = this->atr_parameters[sr.get_param_index()].value();
            sat = this->sat_parameters[sr.get_param_index()].value();
        }
        float gamma{this->gam_parameters[0].value()};

        // Calculate the click probability.
        atr_mul_ex = atr * ex;

        // Calculate the full click probability.
        if (sr.get_click() == 1) {
            full_click_probs.push_back(atr_mul_ex);
        }
        else {
            full_click_probs.push_back(1 - atr_mul_ex);
        }

        ex *= gamma * (1 - atr) + gamma * atr * (1 - sat);
    }
}

/**
 * @brief Frees the memory allocated to the parameters of this click model on
 * the GPU device.
 */
HST void DBN_Hst::destroy_parameters(void) {
    // Free origin and temporary attractiveness containers.
    CUDA_CHECK(hipFree(this->atr_dptr));
    CUDA_CHECK(hipFree(this->atr_tmp_dptr));

    // Free origin and temporary satisfaction containers.
    CUDA_CHECK(hipFree(this->sat_dptr));
    CUDA_CHECK(hipFree(this->sat_tmp_dptr));

    // Free origin and temporary continuation containers.
    CUDA_CHECK(hipFree(this->gam_dptr));
    CUDA_CHECK(hipFree(this->gam_tmp_dptr));

    // Free the device parameter references and sizes.
    CUDA_CHECK(hipFree(this->param_refs));
    CUDA_CHECK(hipFree(this->param_sizes));

    // Reset used device memory.
    this->cm_memory_usage = 0;
}


//---------------------------------------------------------------------------//
// Device-side DBN click model functions.                                    //
//---------------------------------------------------------------------------//

/**
 * @brief Prints a message.
 */
DEV void DBN_Dev::say_hello() {
    printf("Device-side DBN says hello!\n");
}

/**
 * @brief Creates a new DBN click model object.
 *
 * @return The DBN click model object.
 */
DEV DBN_Dev *DBN_Dev::clone() {
    return new DBN_Dev(*this);
}

DEV DBN_Dev::DBN_Dev() = default;

/**
 * @brief Constructs a DBN click model object for the device.
 *
 * @param dbn The base click model object to be copied.
 * @return The DBN click model object.
 */
DEV DBN_Dev::DBN_Dev(DBN_Dev const &dbn) {
}

/**
 * @brief Set the location of the memory allocated for the parameters of this
 * click model on the GPU device.
 *
 * @param parameter_ptr The pointers to the allocated memory.
 * @param parameter_sizes The size of the allocated memory.
 */
DEV void DBN_Dev::set_parameters(Param**& parameter_ptr, int* parameter_sizes) {
    // Set pointers to parameter arrays.
    this->atr_parameters = parameter_ptr[0];
    this->atr_tmp_parameters = parameter_ptr[1];
    this->sat_parameters = parameter_ptr[2];
    this->sat_tmp_parameters = parameter_ptr[3];
    this->gam_parameters = parameter_ptr[4];
    this->gam_tmp_parameters = parameter_ptr[5];

    // Set parameter array sizes.
    this->n_atr_parameters = parameter_sizes[0];
    this->n_atr_tmp_parameters = parameter_sizes[1];
    this->n_sat_parameters = parameter_sizes[2];
    this->n_sat_tmp_parameters = parameter_sizes[3];
    this->n_gam_parameters = parameter_sizes[4];
    this->n_gam_tmp_parameters = parameter_sizes[5];
}

/**
 * @brief Compute a single Expectation-Maximization iteration for the DBN click
 * model, for a single query session.
 *
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 * @param dataset_size The size of the dataset.
 * @param clicks The click on each rank of the query session.
 * @param pidx The parameter index of each rank of the query session.
 */
DEV void DBN_Dev::process_session(SERP_Dev& query_session, int& thread_index, int& dataset_size, const char (&clicks)[BLOCK_SIZE * MAX_SERP], const int (&pidx)[BLOCK_SIZE * MAX_SERP]) {
    int last_click_rank = query_session.last_click_rank();
    float click_probs[MAX_SERP][MAX_SERP] = { 0.f };
    float exam_probs[MAX_SERP + 1];
    float exam[MAX_SERP + 1];
    float car[MAX_SERP + 1] = { 0.f };

    this->gam_tmp_parameters[thread_index].set_values(0.f, 0.f);

    this->compute_exm_car(exam, car, pidx);
    this->compute_dbn_atr(thread_index, last_click_rank, exam, car, dataset_size, clicks, pidx);
    this->compute_dbn_sat(thread_index, last_click_rank, car, dataset_size, clicks, pidx);
    this->get_tail_clicks(click_probs, exam_probs, clicks, pidx);
    this->compute_gamma(thread_index, last_click_rank, click_probs, exam_probs, clicks, pidx);
}

/**
 * @brief Compute the examination parameter for every rank of this query
 * session. The examination parameter can be re-computed every iteration using
 * the values from attractiveness, satisfaction, and continuation parameters
 * from the previous iteration.
 *
 * @param exam The examination parameters for every rank. The first rank is
 * always examined (1).
 * @param car
 * @param pidx The parameter index of each rank of the query session.
 */
DEV void DBN_Dev::compute_exm_car(float (&exam)[MAX_SERP + 1], float (&car)[MAX_SERP + 1], const int (&pidx)[BLOCK_SIZE * MAX_SERP]) {
    // Set the default examination value for the first rank.
    exam[0] = 1.f;

    float attr_val, sat_value, gamma_value, ex_value, temp, car_val;
    float car_helper[MAX_SERP][2];
    int shr_idx;

    for (int rank = 0; rank < MAX_SERP;) {
        shr_idx = pidx[rank * BLOCK_SIZE + threadIdx.x];
        attr_val = this->atr_parameters[shr_idx].value();
        sat_value = this->sat_parameters[shr_idx].value();
        gamma_value = this->gam_parameters[0].value();
        ex_value = exam[rank];

        temp = gamma_value * (1 - attr_val);
        ex_value *= temp + gamma_value * attr_val * (1 - sat_value);

        car_helper[rank][0] = attr_val;
        car_helper[rank][1] = temp;

        rank += 1;
        exam[rank] = ex_value;
    }

    for (int car_itr = MAX_SERP - 1; car_itr > -1; car_itr--) {
        car_val = car[car_itr + 1];

        car[car_itr] = car_helper[car_itr][0] + car_helper[car_itr][1] * car_val;
    }
}

/**
 * @brief Compute the attractiveness parameter for every rank of this query
 * session.
 *
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 * @param last_click_rank The last rank of this query sessions which has been
 * clicked.
 * @param exam The examination parameters for every rank. The first rank is
 * always examined (1).
 * @param car
 * @param dataset_size The size of the dataset.
 * @param clicks The click on each rank of the query session.
 */
DEV void DBN_Dev::compute_dbn_atr(int& thread_index, int& last_click_rank, float (&exam)[MAX_SERP + 1], float (&car)[MAX_SERP + 1], int& dataset_size, const char (&clicks)[BLOCK_SIZE * MAX_SERP], const int (&pidx)[BLOCK_SIZE * MAX_SERP]) {
    float numerator_update, denominator_update;
    float exam_val, attr_val,  car_val;

    #pragma unroll
    for (int rank = 0; rank < MAX_SERP; rank++) {
        numerator_update = 0.f;
        denominator_update = 1.f;

        if (clicks[rank * BLOCK_SIZE + threadIdx.x] == 1) {
            numerator_update += 1.f;
        }
        else if (rank >= last_click_rank) {
            attr_val = this->atr_parameters[pidx[rank * BLOCK_SIZE + threadIdx.x]].value();
            exam_val = exam[rank];
            car_val = car[rank];

            numerator_update += (attr_val * (1 - exam_val)) / (1 - exam_val * car_val);
        }

        this->atr_tmp_parameters[rank * dataset_size + thread_index].set_values(numerator_update, denominator_update);
    }
}

/**
 * @brief Compute the satisfaction parameter for every rank of this query
 * session.
 *
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 * @param last_click_rank The last rank of this query sessions which has been
 * clicked.
 * @param car
 * @param dataset_size The size of the dataset.
 * @param clicks The click on each rank of the query session.
 * @param pidx The parameter index of each rank of the query session.
 */
DEV void DBN_Dev::compute_dbn_sat(int& thread_index, int& last_click_rank, float (&car)[MAX_SERP + 1], int& dataset_size, const char (&clicks)[BLOCK_SIZE * MAX_SERP], const int (&pidx)[BLOCK_SIZE * MAX_SERP]) {
    float numerator_update, denominator_update;
    float gamma_val, sat_val, car_val;

    #pragma unroll
    for (int rank = 0; rank < MAX_SERP; rank++) {
        if (clicks[rank * BLOCK_SIZE + threadIdx.x] == 1) {
            numerator_update = 0.f;
            denominator_update = 1.f;

            if (rank == last_click_rank) {
                sat_val = this->sat_parameters[pidx[rank * BLOCK_SIZE + threadIdx.x]].value();
                gamma_val = this->gam_parameters[0].value();

                if (rank < MAX_SERP - 1) {
                    car_val = car[rank + 1];
                } else{
                    car_val = 0.f;
                }

                numerator_update += sat_val / (1 - (1 - sat_val) * gamma_val * car_val);
            }

            this->sat_tmp_parameters[rank * dataset_size + thread_index].set_values(numerator_update, denominator_update);
        }
    }
}

/**
 * @brief Compute the click probabilities of a rank given the clicks on the
 * preceding ranks.
 *
 * @param click_probs The probabilty of a click occurring on a rank.
 * @param exam_probs The probability of a rank being examined.
 * @param clicks The click on each rank of the query session.
 * @param pidx The parameter index of each rank of the query session.
 */
DEV void DBN_Dev::get_tail_clicks(float (&click_probs)[MAX_SERP][MAX_SERP], float (&exam_probs)[MAX_SERP + 1], const char (&clicks)[BLOCK_SIZE * MAX_SERP], const int (&pidx)[BLOCK_SIZE * MAX_SERP]) {
    exam_probs[0] = 1.f;
    float exam_val, gamma_val, click_prob;

    for (int start_rank = 0; start_rank < MAX_SERP; start_rank++) {
        exam_val = 1.f;

        int ses_itr{0};
        for (int res_itr = start_rank; res_itr < MAX_SERP; res_itr++) {
            int shr_idx = pidx[ses_itr * BLOCK_SIZE + threadIdx.x];
            float attr_val = this->atr_parameters[shr_idx].value();
            float sat_val = this->sat_parameters[shr_idx].value();
            gamma_val = this->gam_parameters[0].value();

            if (clicks[res_itr * BLOCK_SIZE + threadIdx.x] == 1){
                click_prob = attr_val * exam_val;
                exam_val = gamma_val * (1 - sat_val);
            }
            else{
                click_prob = 1 - attr_val * exam_val;
                exam_val *= gamma_val * (1 - attr_val) / click_prob;
            }

            click_probs[start_rank][ses_itr] = click_prob;

            if (start_rank == 0) {
                exam_probs[ses_itr + 1] = exam_val;
            }

            ses_itr++;
        }
    }
}

/**
 * @brief Compute the continuation parameter gamma.
 *
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 * @param last_click_rank The last rank of this query sessions which has been
 * clicked.
 * @param click_probs The probabilty of a click occurring on a rank.
 * @param exam_probs The probability of a rank being examined.
 * @param clicks The click on each rank of the query session.
 * @param pidx The parameter index of each rank of the query session.
 */
DEV void DBN_Dev::compute_gamma(int& thread_index, int& last_click_rank, float (&click_probs)[MAX_SERP][MAX_SERP], float (&exam_probs)[MAX_SERP + 1], const char (&clicks)[BLOCK_SIZE * MAX_SERP], const int (&pidx)[BLOCK_SIZE * MAX_SERP]) {
    float factor_values[8] = { 0.f };

    #pragma unroll
    for (int rank = 0; rank < MAX_SERP; rank++){
        // Send the initialization values to the phi function.
        int shr_idx = pidx[rank * BLOCK_SIZE + threadIdx.x];
        DBNFactor factor_func(click_probs, exam_probs, clicks[rank * BLOCK_SIZE + threadIdx.x],
                              last_click_rank, rank,
                              this->atr_parameters[shr_idx].value(),
                              this->sat_parameters[shr_idx].value(),
                              this->gam_parameters[0].value());

        float factor_result = 0.f;
        float factor_sum = 0.f;

        // Compute phi for all possible input values.
        for (int fct_itr{0}; fct_itr < 8; fct_itr++) {
            factor_result = factor_func.compute(this->factor_inputs[fct_itr][0],
                                                this->factor_inputs[fct_itr][1],
                                                this->factor_inputs[fct_itr][2]);
            factor_values[fct_itr] = factor_result;
            factor_sum += factor_result;
        }

        float numerator_update = factor_values[5] / factor_sum;
        float denominator_update = (factor_values[4] + factor_values[5]) / factor_sum;

        this->gam_tmp_parameters[thread_index].add_to_values(numerator_update, denominator_update);
    }
}

/**
 * @brief Update the global parameter values using the local parameter values
 * on each thread.
 *
 * @param thread_index The global index of the thread.
 * @param block_index The index of the block in which this thread exists.
 * @param dataset_size The size of the dataset.
 * @param pidx The unique parameter index of each rank of the query session.
 */
DEV void DBN_Dev::update_parameters(int& thread_index, int& block_index, int& dataset_size, const int (&pidx)[BLOCK_SIZE * MAX_SERP]) {
    update_shared_parameters_dev(this->gam_tmp_parameters, this->gam_parameters, thread_index, this->n_gam_parameters, block_index, dataset_size);

    if (thread_index < dataset_size) {
        update_unique_parameters_dev(this->atr_tmp_parameters, this->atr_parameters, thread_index, dataset_size, pidx);
        update_unique_parameters_dev(this->sat_tmp_parameters, this->sat_parameters, thread_index, dataset_size, pidx);
    }
}
