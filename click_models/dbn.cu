#include "hip/hip_runtime.h"
/** DBN click model.
 * Pooya Khandel's ParClick is used as a reference implementation.
 *
 * dbn.cu:
 *  - Defines the functions specific to creating a DBN CM.
 */

#include "dbn.cuh"


//---------------------------------------------------------------------------//
// Host-side DBN click model functions.                                      //
//---------------------------------------------------------------------------//

HST DBN_Host::DBN_Host() = default;

/**
 * @brief Constructs a DBN click model object for the host.
 *
 * @param dbn
 * @returns DBN_Host The DBN click model object.
 */
HST DBN_Host::DBN_Host(DBN_Host const &dbn) {
}

/**
 * @brief Creates a new DBN click model object.
 *
 * @return DBN_Host* The DBN click model object.
 */
HST DBN_Host* DBN_Host::clone() {
    return new DBN_Host(*this);
}

/**
 * @brief Print a message.
 */
HST void DBN_Host::say_hello() {
    std::cout << "Host-side DBN says hello!" << std::endl;
}

/**
 * @brief Get the amount of device memory allocated to this click model.
 *
 * @return size_t The used memory.
 */
HST size_t DBN_Host::get_memory_usage(void) {
    return this->cm_memory_usage;
}

/**
 * @brief Allocate device-side memory for the attractiveness parameters.
 *
 * @param partition The training and testing sets, and the number of
 * query-document pairs in the training set.
 * @param n_devices The number of devices on this node.
 */
HST void DBN_Host::init_attractiveness_parameters(const std::tuple<std::vector<SERP>, std::vector<SERP>, int>& partition, int n_devices) {
    Param default_parameter;
    default_parameter.set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);

    // Allocate memory for the attractiveness parameters on the device.
    this->n_attr_dev = std::get<2>(partition);
    this->attractiveness_parameters.resize(this->n_attr_dev, default_parameter);
    CUDA_CHECK(hipMalloc(&this->attr_param_dptr, this->n_attr_dev * sizeof(Param)));
    CUDA_CHECK(hipMemcpy(this->attr_param_dptr, this->attractiveness_parameters.data(),
                          this->n_attr_dev * sizeof(Param), hipMemcpyHostToDevice));

    // Allocate memory for the temporary attractiveness parameters on the device.
    // These values are replaced at the start of each iteration, which means
    // they don't need to be initialized with a CUDA memory copy.
    this->n_tmp_attr_dev = std::get<0>(partition).size() * MAX_SERP_LENGTH;
    this->tmp_attractiveness_parameters.resize(this->n_tmp_attr_dev);
    CUDA_CHECK(hipMalloc(&this->tmp_attr_param_dptr, this->n_tmp_attr_dev * sizeof(Param)));

    // Store the number of allocated bytes.
    this->cm_memory_usage += this->n_attr_dev * sizeof(Param) + this->n_tmp_attr_dev * sizeof(Param);
}

/**
 * @brief Allocate device-side memory for the satisfaction parameters.
 *
 * @param partition The training and testing sets, and the number of
 * query-document pairs in the training set.
 * @param n_devices The number of devices on this node.
 */
HST void DBN_Host::init_satisfaction_parameters(const std::tuple<std::vector<SERP>, std::vector<SERP>, int>& partition, int n_devices) {
    Param default_parameter;
    default_parameter.set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);

    // Allocate memory for the satisfaction parameters on the device.
    this->n_satisfaction_dev = std::get<2>(partition);
    this->satisfaction_parameters.resize(this->n_satisfaction_dev, default_parameter);
    CUDA_CHECK(hipMalloc(&this->satisfaction_param_dptr, this->n_satisfaction_dev * sizeof(Param)));
    CUDA_CHECK(hipMemcpy(this->satisfaction_param_dptr, this->satisfaction_parameters.data(),
                          this->n_satisfaction_dev * sizeof(Param), hipMemcpyHostToDevice));

    // Allocate memory for the temporary satisfaction parameters on the device.
    // These values are replaced at the start of each iteration, which means
    // they don't need to be initialized with a CUDA memory copy.
    this->n_tmp_satisfaction_dev = std::get<0>(partition).size() * MAX_SERP_LENGTH;
    this->tmp_satisfaction_parameters.resize(this->n_tmp_satisfaction_dev);
    CUDA_CHECK(hipMalloc(&this->tmp_satisfaction_param_dptr, this->n_tmp_satisfaction_dev * sizeof(Param)));

    // Store the number of allocated bytes.
    this->cm_memory_usage += this->n_satisfaction_dev * sizeof(Param) + this->n_tmp_satisfaction_dev * sizeof(Param);
}

/**
 * @brief Allocate device-side memory for the continuation parameters, gamma.
 *
 * @param partition The training and testing sets, and the number of
 * query-document pairs in the training set.
 * @param n_devices The number of devices on this node.
 */
HST void DBN_Host::init_gamma_parameters(const std::tuple<std::vector<SERP>, std::vector<SERP>, int>& partition, int n_devices) {
    Param default_parameter;
    default_parameter.set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);

    // Allocate memory for the continuation parameters on the device.
    this->n_gamma_dev = 1;
    this->gamma_parameters.resize(this->n_gamma_dev, default_parameter);
    CUDA_CHECK(hipMalloc(&this->gamma_param_dptr, this->n_gamma_dev * sizeof(Param)));
    CUDA_CHECK(hipMemcpy(this->gamma_param_dptr, this->gamma_parameters.data(),
                          this->n_gamma_dev * sizeof(Param), hipMemcpyHostToDevice));

    // Allocate memory for the temporary continuation parameters on the device.
    // These values are replaced at the start of each iteration, which means
    // they don't need to be initialized with a CUDA memory copy.
    this->n_tmp_gamma_dev = std::get<0>(partition).size() * this->n_gamma_dev;
    this->tmp_gamma_parameters.resize(this->n_tmp_gamma_dev);
    CUDA_CHECK(hipMalloc(&this->tmp_gamma_param_dptr, this->n_tmp_gamma_dev * sizeof(Param)));

    // Store the number of allocated bytes.
    this->cm_memory_usage += this->n_gamma_dev * sizeof(Param) + this->n_tmp_gamma_dev * sizeof(Param);
}

/**
 * @brief Allocate device-side memory for the attractiveness, satisfaction and
 * continuation parameters of the click model.
 *
 * @param partition The training and testing sets, and the number of
 * query-document pairs in the training set.
 * @param n_devices The number of devices on this node.
 */
HST void DBN_Host::init_parameters(const std::tuple<std::vector<SERP>, std::vector<SERP>, int>& partition, int n_devices) {
    this->init_attractiveness_parameters(partition, n_devices);
    this->init_satisfaction_parameters(partition, n_devices);
    this->init_gamma_parameters(partition, n_devices);
}

/**
 * @brief Get the references to the allocated device-side memory.
 *
 * @param param_refs An array containing the references to the device-side
 * parameters in memory.
 * @param param_sizes The size of each of the memory allocations on the device.
 */
HST void DBN_Host::get_device_references(Param**& param_refs, int*& param_sizes) {
    int n_references = 6;

    // Create a temporary array to store the device references.
    Param* tmp_param_refs_array[n_references];
    tmp_param_refs_array[0] = this->attr_param_dptr;
    tmp_param_refs_array[1] = this->tmp_attr_param_dptr;
    tmp_param_refs_array[2] = this->satisfaction_param_dptr;
    tmp_param_refs_array[3] = this->tmp_satisfaction_param_dptr;
    tmp_param_refs_array[4] = this->gamma_param_dptr;
    tmp_param_refs_array[5] = this->tmp_gamma_param_dptr;

    // Allocate space for the device references.
    CUDA_CHECK(hipMalloc(&param_refs, n_references * sizeof(Param*)));
    CUDA_CHECK(hipMemcpy(param_refs, tmp_param_refs_array,
                          n_references * sizeof(Param*), hipMemcpyHostToDevice));

    int tmp_param_sizes_array[n_references];
    tmp_param_sizes_array[0] = this->n_attr_dev;
    tmp_param_sizes_array[1] = this->n_tmp_attr_dev;
    tmp_param_sizes_array[2] = this->n_satisfaction_dev;
    tmp_param_sizes_array[3] = this->n_tmp_satisfaction_dev;
    tmp_param_sizes_array[4] = this->n_gamma_dev;
    tmp_param_sizes_array[5] = this->n_tmp_gamma_dev;

    // Allocate space for the device references.
    CUDA_CHECK(hipMalloc(&param_sizes, n_references * sizeof(int)));
    CUDA_CHECK(hipMemcpy(param_sizes, tmp_param_sizes_array,
                          n_references * sizeof(int), hipMemcpyHostToDevice));

    // Keep track of the pointers to the allocated device-side memory.
    this->param_refs = param_refs;
    this->param_sizes = param_sizes;
    this->cm_memory_usage += n_references * sizeof(Param*) + n_references * sizeof(int);
}

/**
 * @brief Update the global parameter values with the temporarily stored new
 * local parameter values on each thread.
 *
 * @param gridSize The size of kernel blocks on the GPU.
 * @param blockSize The number of kernel threads per block on the GPU.
 * @param partition The dataset allocated on the GPU.
 * @param dataset_size The size of the allocated dataset.
 */
HST void DBN_Host::update_parameters(int& gridSize, int& blockSize, SERP*& partition, int& dataset_size) {
    Kernel::update<<<gridSize, blockSize>>>(partition, dataset_size, 0);
}

/**
 * @brief Reset the original parameter values to zero so the previous parameter
 * values won't affect the next result twice.
 *
 * Further explanation; The first time it would affect the result would be when
 * retrieving their values in the training kernel, and the (unnecessary) second
 * time would be when adding the values to the original parameter containers.
 * The second time would still give a valid result but would slow down the
 * converging of the parameters.
 */
HST void DBN_Host::reset_parameters(void) {
    // Create a parameter initialized at 0.
    Param default_parameter;
    default_parameter.set_values(PARAM_DEF_NUM, PARAM_DEF_DENOM);

    // Create an array of the right proportions with the empty parameters.
    std::vector<Param> cleared_attractiveness_parameters(this->n_attr_dev, default_parameter);
    std::vector<Param> cleared_satisfaction_parameters(this->n_satisfaction_dev, default_parameter);
    std::vector<Param> cleared_gamma_parameters(this->n_gamma_dev, default_parameter);

    // Copy the cleared array to the device.
    CUDA_CHECK(hipMemcpy(this->attr_param_dptr, cleared_attractiveness_parameters.data(), this->n_attr_dev * sizeof(Param), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(this->satisfaction_param_dptr, cleared_satisfaction_parameters.data(), this->n_satisfaction_dev * sizeof(Param), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(this->gamma_param_dptr, cleared_gamma_parameters.data(), this->n_gamma_dev * sizeof(Param), hipMemcpyHostToDevice));
}

/**
 * @brief Transfers parameters of a given type either from the device to the
 * host, or the otherway around.
 *
 * @param parameter_type The type of parameter that will be transfered.
 * (PUBLIC, PRIVATE, or ALL).
 * @param transfer_direction The direction in which the transfer will happen.
 * (H2D or D2H).
 */
HST void DBN_Host::transfer_parameters(int parameter_type, int transfer_direction) {
    // Public parameters.
    if (parameter_type == PUBLIC || parameter_type == ALL) {
        if (transfer_direction == D2H) { // Transfer from device to host.
            // Retrieve the continuation parameters from the device.
            CUDA_CHECK(hipMemcpy(this->gamma_parameters.data(), this->gamma_param_dptr, this->n_gamma_dev * sizeof(Param), hipMemcpyDeviceToHost));
        }
        else if (transfer_direction == H2D) { // Transfer from host to device.
            // Send the continuation parameters to the device.
            CUDA_CHECK(hipMemcpy(this->gamma_param_dptr, this->gamma_parameters.data(), this->n_gamma_dev * sizeof(Param), hipMemcpyHostToDevice));
        }
    }

    // Private parameters.
    if (parameter_type == PRIVATE || parameter_type == ALL) {
        if (transfer_direction == D2H) { // Transfer from device to host.
            // Retrieve the attractiveness parameters from the device.
            CUDA_CHECK(hipMemcpy(this->attractiveness_parameters.data(), this->attr_param_dptr, this->n_attr_dev * sizeof(Param), hipMemcpyDeviceToHost));
        }
        else if (transfer_direction == H2D) { // Transfer from host to device.
            // Send the attractiveness parameters to the device.
            CUDA_CHECK(hipMemcpy(this->attr_param_dptr, this->attractiveness_parameters.data(), this->n_attr_dev * sizeof(Param), hipMemcpyHostToDevice));
        }

        if (transfer_direction == D2H) { // Transfer from device to host.
            // Retrieve the satisfaction parameters from the device.
            CUDA_CHECK(hipMemcpy(this->satisfaction_parameters.data(), this->satisfaction_param_dptr, this->n_satisfaction_dev * sizeof(Param), hipMemcpyDeviceToHost));
        }
        else if (transfer_direction == H2D) { // Transfer from host to device.
            // Send the satisfaction parameters to the device.
            CUDA_CHECK(hipMemcpy(this->satisfaction_param_dptr, this->satisfaction_parameters.data(), this->n_satisfaction_dev * sizeof(Param), hipMemcpyHostToDevice));
        }
    }
}

/**
 * @brief Retrieve the parameters of a given type into a given array from the
 * click model.
 *
 * @param destination The array which will save the indicated parameters.
 * @param parameter_type The type of parameters which will be retrieved
 * (PUBLIC, PRIVATE, or ALL).
 */
HST void DBN_Host::get_parameters(std::vector<std::vector<Param>>& destination, int parameter_type) {
    // Add the parameters to a generic vector which can represent  multiple
    // retrieved parameter types.
    if (parameter_type == PUBLIC) {
        destination.resize(1);
        destination[0] = this->gamma_parameters;
    }
    else if (parameter_type == PRIVATE) {
        destination.resize(2);
        destination[0] = this->attractiveness_parameters;
        destination[1] = this->satisfaction_parameters;
    }
    else if (parameter_type == ALL) {
        destination.resize(3);
        destination[0] = this->attractiveness_parameters;
        destination[1] = this->satisfaction_parameters;
        destination[2] = this->gamma_parameters;
    }
}

/**
 * @brief Compute the result of combining the DBN parameters from other nodes
 * or devices.
 *
 * @param parameters A multi-dimensional vector containing the parameters to be
 * combined. The vector is structured as follows: Node/Device ID -> Parameter
 * type -> Parameters.
 */
HST void DBN_Host::sync_parameters(std::vector<std::vector<std::vector<Param>>>& parameters) {
    for (int r = 0; r < parameters[0][0].size(); r++) {
        for (int param_type = 0; param_type < parameters[0].size(); param_type++) {
            // Use the first sub-array to combine the results in.
            Param base = parameters[0][param_type][r];

            // Subtract the starting values of other partitions.
            parameters[0][param_type][r].set_values(base.numerator_val() - (parameters.size() - 1),
                                                    base.denominator_val() - 2 * (parameters.size() - 1));

            for (int device_id = 1; device_id < parameters.size(); device_id++) {
                Param ex = parameters[device_id][param_type][r];
                parameters[0][param_type][r].add_to_values(ex.numerator_val(),
                                                           ex.denominator_val());
            }
        }
    }
}

/**
 * @brief Set the parameters of a host-side click model equal to the given
 * given arguments.
 *
 * @param source The new parameter values.
 * @param parameter_type The type of the given parameters. (PUBLIC, PRIVATE, or
 * ALL).
 */
HST void DBN_Host::set_parameters(std::vector<std::vector<Param>>& source, int parameter_type) {
    // Set the parameters of this click model.
    if (parameter_type == PUBLIC) {
        this->gamma_parameters = source[0];
    }
    else if (parameter_type == PRIVATE) {
        this->attractiveness_parameters = source[0];
        this->satisfaction_parameters = source[1];
    }
    else if (parameter_type == ALL) {
        this->attractiveness_parameters = source[0];
        this->satisfaction_parameters = source[1];
        this->gamma_parameters = source[2];
    }
}

/**
 * @brief Compute the log-likelihood of the current DBN for the given query
 * session.
 *
 * @param query_session The query session for which the log-likelihood will be
 * computed.
 * @param log_click_probs The vector which will store the log-likelihood for
 * the document at each rank in the query session.
 */
HST void DBN_Host::get_log_conditional_click_probs(SERP& query_session, std::vector<float>& log_click_probs) {
    float ex{1.f}, click_prob;

    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        SearchResult sr = query_session[rank];

        // Get the parameters corresponding to the current search result.
        // Return the default parameter value if the qd-pair was not found in
        // the training set.
        float attr_val{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        float sat_val{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        if (sr.get_param_index() != -1) {
            attr_val = this->attractiveness_parameters[sr.get_param_index()].value();
            sat_val = this->satisfaction_parameters[sr.get_param_index()].value();
        }
        float gamma_val{this->gamma_parameters[0].value()};

        if (sr.get_click() == 1) {
            click_prob = attr_val * ex;
            ex = gamma_val * ( 1- sat_val);
        } else{
            click_prob = 1 - attr_val * ex;
            ex *= gamma_val * ( 1 - attr_val) / click_prob;
        }

        log_click_probs.push_back(std::log(click_prob));
    }
}

/**
 * @brief Compute the click probability of the current DBN for the given query
 * session.
 *
 * @param query_session The query session for which the click probability will
 * be computed.
 * @param full_click_probs The vector which will store the click probability
 * for the document at each rank in the query session.
 */
HST void DBN_Host::get_full_click_probs(SERP& query_session, std::vector<float> &full_click_probs) {
    float ex{1.f}, atr_mul_ex;

    // Go through all ranks of the query session.
    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        // Retrieve the search result at the current rank.
        SearchResult sr = query_session[rank];

        // Get the parameters corresponding to the current search result.
        // Return the default parameter value if the qd-pair was not found in
        // the training set.
        float atr{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        float sat{(float) PARAM_DEF_NUM / (float) PARAM_DEF_DENOM};
        if (sr.get_param_index() != -1) {
            atr = this->attractiveness_parameters[sr.get_param_index()].value();
            sat = this->satisfaction_parameters[sr.get_param_index()].value();
        }
        float gamma{this->gamma_parameters[0].value()};

        // Calculate the click probability.
        atr_mul_ex = atr * ex;

        // Calculate the full click probability.
        if (sr.get_click() == 1) {
            full_click_probs.push_back(atr_mul_ex);
        }
        else {
            full_click_probs.push_back(1 - atr_mul_ex);
        }

        ex *= gamma * (1 - atr) + gamma * atr * (1 - sat);
    }
}

/**
 * @brief Frees the memory allocated to the parameters of this click model on
 * the GPU device.
 */
HST void DBN_Host::destroy_parameters(void) {
    // Free origin and temporary attractiveness containers.
    CUDA_CHECK(hipFree(this->attr_param_dptr));
    CUDA_CHECK(hipFree(this->tmp_attr_param_dptr));

    // Free origin and temporary satisfaction containers.
    CUDA_CHECK(hipFree(this->satisfaction_param_dptr));
    CUDA_CHECK(hipFree(this->tmp_satisfaction_param_dptr));

    // Free origin and temporary continuation containers.
    CUDA_CHECK(hipFree(this->gamma_param_dptr));
    CUDA_CHECK(hipFree(this->tmp_gamma_param_dptr));

    // Free the device parameter references and sizes.
    CUDA_CHECK(hipFree(this->param_refs));
    CUDA_CHECK(hipFree(this->param_sizes));

    // Reset used device memory.
    this->cm_memory_usage = 0;
}


//---------------------------------------------------------------------------//
// Device-side DBN click model functions.                                    //
//---------------------------------------------------------------------------//

/**
 * @brief Prints a message.
 */
DEV void DBN_Dev::say_hello() {
    printf("Device-side DBN says hello!\n");
}

/**
 * @brief Creates a new DBN click model object.
 *
 * @return DBN_Dev* The DBN click model object.
 */
DEV DBN_Dev *DBN_Dev::clone() {
    return new DBN_Dev(*this);
}

DEV DBN_Dev::DBN_Dev() = default;

/**
 * @brief Constructs a DBN click model object for the device.
 *
 * @param dbn
 * @returns DBN_Dev The DBN click model object.
 */
DEV DBN_Dev::DBN_Dev(DBN_Dev const &dbn) {
}

/**
 * @brief Set the location of the memory allocated for the parameters of this
 * click model on the GPU device.
 *
 * @param parameter_ptr The pointers to the allocated memory.
 * @param parameter_sizes The size of the allocated memory.
 */
DEV void DBN_Dev::set_parameters(Param**& parameter_ptr, int* parameter_sizes) {
    // Set pointers to parameter arrays.
    this->attractiveness_parameters = parameter_ptr[0];
    this->tmp_attractiveness_parameters = parameter_ptr[1];
    this->satisfaction_parameters = parameter_ptr[2];
    this->tmp_satisfaction_parameters = parameter_ptr[3];
    this->gamma_parameters = parameter_ptr[4];
    this->tmp_gamma_parameters = parameter_ptr[5];

    // Set parameter array sizes.
    this->n_attractiveness_parameters = parameter_sizes[0];
    this->n_tmp_attractiveness_parameters = parameter_sizes[1];
    this->n_satisfaction_parameters = parameter_sizes[2];
    this->n_tmp_satisfaction_parameters = parameter_sizes[3];
    this->n_gamma_parameters = parameter_sizes[4];
    this->n_tmp_gamma_parameters = parameter_sizes[5];
}

/**
 * @brief Compute a single Expectation-Maximization iteration for the DBN click
 * model, for a single query session.
 *
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 */
DEV void DBN_Dev::process_session(SERP& query_session, int& thread_index) {
    int last_click_rank = query_session.last_click_rank();
    float click_probs[MAX_SERP_LENGTH][MAX_SERP_LENGTH] = { 0.f };
    float exam_probs[MAX_SERP_LENGTH + 1];
    float exam[MAX_SERP_LENGTH + 1];
    float car[MAX_SERP_LENGTH + 1] = { 0.f };

    this->tmp_gamma_parameters[thread_index].set_values(0.f, 0.f);

    this->compute_exam_car(thread_index, query_session, exam, car);
    this->compute_dbn_attr(thread_index, query_session, last_click_rank, exam, car);
    this->compute_dbn_sat(thread_index, query_session, last_click_rank, car);
    this->get_tail_clicks(thread_index, query_session, click_probs, exam_probs);
    this->compute_gamma(thread_index, query_session, last_click_rank, click_probs, exam_probs);
}

/**
 * @brief Compute the examination parameter for every rank of this query
 * session. The examination parameter can be re-computed every iteration using
 * the values from attractiveness, satisfaction, and continuation parameters
 * from the previous iteration.
 *
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param exam The examination parameters for every rank. The first rank is
 * always examined (1).
 * @param car
 */
DEV void DBN_Dev::compute_exam_car(int& thread_index, SERP& query_session, float (&exam)[MAX_SERP_LENGTH + 1], float (&car)[MAX_SERP_LENGTH + 1]) {
    // Set the default examination value for the first rank.
    exam[0] = 1.f;

    float attr_val, sat_value, gamma_value, ex_value, temp, car_val;
    float car_helper[MAX_SERP_LENGTH][2];

    for (int rank = 0; rank < MAX_SERP_LENGTH;) {
        SearchResult sr = query_session[rank];

        attr_val = this->attractiveness_parameters[sr.get_param_index()].value();
        sat_value = this->satisfaction_parameters[sr.get_param_index()].value();
        gamma_value = this->gamma_parameters[0].value();
        ex_value = exam[rank];

        temp = gamma_value * (1 - attr_val);
        ex_value *= temp + gamma_value * attr_val * (1 - sat_value);

        car_helper[rank][0] = attr_val;
        car_helper[rank][1] = temp;

        rank += 1;
        exam[rank] = ex_value;
    }

    for (int car_itr = MAX_SERP_LENGTH - 1; car_itr > -1; car_itr--) {
        car_val = car[car_itr + 1];

        car[car_itr] = car_helper[car_itr][0] + car_helper[car_itr][1] * car_val;
    }
}

/**
 * @brief Compute the attractiveness parameter for every rank of this query
 * session.
 *
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param last_click_rank The last rank of this query sessions which has been
 * clicked.
 * @param exam The examination parameters for every rank. The first rank is
 * always examined (1).
 * @param car
 */
DEV void DBN_Dev::compute_dbn_attr(int& thread_index, SERP& query_session, int& last_click_rank, float (&exam)[MAX_SERP_LENGTH + 1], float (&car)[MAX_SERP_LENGTH + 1]) {
    float numerator_update, denominator_update;
    float exam_val, attr_val,  car_val;

    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        SearchResult sr = query_session[rank];

        numerator_update = 0.f;
        denominator_update = 1.f;

        if (sr.get_click() == 1) {
            numerator_update += 1.f;
        }
        else if (rank >= last_click_rank) {
            attr_val = this->attractiveness_parameters[sr.get_param_index()].value();
            exam_val = exam[rank];
            car_val = car[rank];

            numerator_update += (attr_val * (1 - exam_val)) / (1 - exam_val * car_val);
        }

        this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].set_values(numerator_update, denominator_update);
    }
}

/**
 * @brief Compute the satisfaction parameter for every rank of this query
 * session.
 *
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param last_click_rank The last rank of this query sessions which has been
 * clicked.
 * @param car
 */
DEV void DBN_Dev::compute_dbn_sat(int& thread_index, SERP& query_session, int& last_click_rank, float (&car)[MAX_SERP_LENGTH + 1]) {
    float numerator_update, denominator_update;
    float gamma_val, sat_val, car_val;

    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        SearchResult sr = query_session[rank];

        if (sr.get_click() == 1) {
            numerator_update = 0.f;
            denominator_update = 1.f;

            if (rank == last_click_rank){
                sat_val = this->satisfaction_parameters[sr.get_param_index()].value();
                gamma_val = this->gamma_parameters[0].value();

                if (rank < MAX_SERP_LENGTH - 1) {
                    car_val = car[rank + 1];
                } else{
                    car_val = 0.f;
                }

                numerator_update += sat_val / (1 - (1 - sat_val) * gamma_val * car_val);
            }

            this->tmp_satisfaction_parameters[thread_index * MAX_SERP_LENGTH + rank].set_values(numerator_update, denominator_update);
        }
    }
}

/**
 * @brief Compute the click probabilities of a rank given the clicks on the
 * preceding ranks.
 *
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param click_probs The probabilty of a click occurring on a rank.
 * @param exam_probs The probability of a rank being examined.
 */
DEV void DBN_Dev::get_tail_clicks(int& thread_index, SERP& query_session, float (&click_probs)[MAX_SERP_LENGTH][MAX_SERP_LENGTH], float (&exam_probs)[MAX_SERP_LENGTH + 1]) {
    exam_probs[0] = 1.f;
    float exam_val, gamma_val, click_prob;

    for (int start_rank = 0; start_rank < MAX_SERP_LENGTH; start_rank++) {
        exam_val = 1.f;

        int ses_itr{0};
        for (int res_itr = start_rank; res_itr < MAX_SERP_LENGTH; res_itr++, ses_itr++) {
            SearchResult tmp_sr = query_session[ses_itr];

            float attr_val = this->attractiveness_parameters[tmp_sr.get_param_index()].value();
            float sat_val = this->satisfaction_parameters[tmp_sr.get_param_index()].value();
            gamma_val = this->gamma_parameters[0].value();

            if (query_session[res_itr].get_click() == 1){
                click_prob = attr_val * exam_val;
                exam_val = gamma_val * (1 - sat_val);
            }
            else{
                click_prob = 1 - attr_val * exam_val;
                exam_val *= gamma_val * (1 - attr_val) / click_prob;
            }

            click_probs[start_rank][ses_itr] = click_prob;

            if (start_rank == 0) {
                exam_probs[ses_itr + 1] = exam_val;
            }

            ses_itr++;
        }
    }
}

/**
 * @brief Compute the continuation parameter gamma.
 *
 * @param thread_index The index of the thread which will be estimating the
 * parameters.
 * @param query_session The query session which will be used to estimate the
 * DBN parameters.
 * @param last_click_rank The last rank of this query sessions which has been
 * clicked.
 * @param click_probs The probabilty of a click occurring on a rank.
 * @param exam_probs The probability of a rank being examined.
 */
DEV void DBN_Dev::compute_gamma(int& thread_index, SERP& query_session, int& last_click_rank, float (&click_probs)[MAX_SERP_LENGTH][MAX_SERP_LENGTH], float (&exam_probs)[MAX_SERP_LENGTH + 1]) {
    float factor_values[8] = { 0.f };

    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++){
        SearchResult sr = query_session[rank];

        // Send the initialization values to the phi function.
        DBNFactor factor_func(click_probs, exam_probs, sr.get_click(),
                              last_click_rank, rank,
                              this->attractiveness_parameters[sr.get_param_index()].value(),
                              this->satisfaction_parameters[sr.get_param_index()].value(),
                              this->gamma_parameters[0].value());

        float factor_result = 0.f;
        float factor_sum = 0.f;

        // Compute phi for all possible input values.
        for (int fct_itr{0}; fct_itr < 8; fct_itr++) {
            factor_result = factor_func.compute(this->factor_inputs[fct_itr][0],
                                                this->factor_inputs[fct_itr][1],
                                                this->factor_inputs[fct_itr][2]);
            factor_values[fct_itr] = factor_result;
            factor_sum += factor_result;
        }

        float numerator_update = factor_values[5] / factor_sum;
        float denominator_update = (factor_values[4] + factor_values[5]) / factor_sum;

        this->tmp_gamma_parameters[thread_index].add_to_values(numerator_update, denominator_update);
    }
}



/**
 * @brief Update the global parameter values using the local parameter values
 * on each thread.
 *
 * @param query_session The query session of this thread.
 * @param thread_index The index of the thread.
 * @param block_index The index of the block in which this thread exists.
 * @param parameter_type The type of parameter to update.
 * @param partition_size The size of the dataset.
 */
DEV void DBN_Dev::update_parameters(SERP& query_session, int& thread_index, int& block_index, int& parameter_type, int& partition_size) {
    this->update_gamma_parameters(query_session, thread_index, block_index, partition_size);

    if (thread_index < partition_size) {
        this->update_attractiveness_parameters(query_session, thread_index);
        this->update_satisfaction_parameters(query_session, thread_index);
    }
}


/**
 * @brief Update the global continuation parameters using the local continuation
 * parameters of a single thread.
 *
 * @param query_session The query session of this thread.
 * @param thread_index The index of this thread.
 * @param block_index The index of the block in which this thread exists.
 * @param partition_size The size of the dataset.
 */
DEV void DBN_Dev::update_gamma_parameters(SERP& query_session, int& thread_index, int& block_index, int& partition_size) {
    // Initialize shared memory for this block's continuation parameters at 0.
    SHR float block_gamma_num;
    SHR float block_gamma_denom;
    block_gamma_num = 0.f;
    block_gamma_denom = 0.f;
    // Wait for all threads to finish initializing shared memory.
    __syncthreads();

    // Atomically add the values of the continuation parameters of this thread's
    // query session to the shared continuation parameters of this block.
    // Start every thread in this block at a different query session ranks
    // so prevent all threads from atomically writing to the same rank at the
    // same time.
    if (thread_index < partition_size) {
        // Atomically add the numerator and denominator values to shared memory.
        atomicAddArch(&block_gamma_num, this->tmp_gamma_parameters[thread_index].numerator_val());
        atomicAddArch(&block_gamma_denom, this->tmp_gamma_parameters[thread_index].denominator_val());
    }
    // Wait for all threads to finish writing to shared memory.
    __syncthreads();

    // Have only the first thread of the block write the shared memory
    // results to global memory.
    if (block_index == 0) {
        this->gamma_parameters[block_index].atomic_add_to_values(block_gamma_num, block_gamma_denom);
    }
}

/**
 * @brief Update the global attractiveness parameters using the local
 * attractiveness parameters of a single thread.
 *
 * @param query_session The query session of this thread.
 * @param thread_index The index of this thread.
 */
DEV void DBN_Dev::update_attractiveness_parameters(SERP& query_session, int& thread_index) {
    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        SearchResult sr = query_session[rank];
        this->attractiveness_parameters[sr.get_param_index()].atomic_add_to_values(
            this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].numerator_val(),
            this->tmp_attractiveness_parameters[thread_index * MAX_SERP_LENGTH + rank].denominator_val());
    }
}

/**
 * @brief Update the global satisfaction parameters using the local satisfaction
 * parameters of a single thread.
 *
 * @param query_session The query session of this thread.
 * @param thread_index The index of this thread.
 * @param block_index The index of the block in which this thread exists.
 * @param partition_size The size of the dataset.
 */
DEV void DBN_Dev::update_satisfaction_parameters(SERP& query_session, int& thread_index) {
    for (int rank = 0; rank < MAX_SERP_LENGTH; rank++) {
        SearchResult sr = query_session[rank];
        this->satisfaction_parameters[sr.get_param_index()].atomic_add_to_values(
            this->tmp_satisfaction_parameters[thread_index * MAX_SERP_LENGTH + rank].numerator_val(),
            this->tmp_satisfaction_parameters[thread_index * MAX_SERP_LENGTH + rank].denominator_val());
    }
}