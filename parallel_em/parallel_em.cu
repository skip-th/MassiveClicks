#include "hip/hip_runtime.h"
/** Parallelizing EM on GPU(s).
 * Pooya Khandel's ParClick is used as a reference implementation.
 *
 * parallel_em.cu:
 *  - Defines the functions used for initiating the EM process on the GPU.
 */

// User include.
#include "parallel_em.cuh"


//---------------------------------------------------------------------------//
// Host-side computation.                                                    //
//---------------------------------------------------------------------------//

/**
 * @brief Runs the Expectation-Maximization algorithm for a given click model
 * on one or multiple GPU's and evaluates the result.
 *
 * @param model_type The type of click model (e.g. 0 = PBM).
 * @param node_id The MPI communication rank of this node.
 * @param n_nodes The number of nodes in the network.
 * @param n_devices_network The number of devices per node in the network.
 * @param n_itr The number of iterations for which the EM algorithm should run.
 * @param device_partitions The training and testing sets and the number of query
 * document pairs in the training set, for each device on this node.
 */
void em_parallel(const int model_type, const int node_id, const int n_nodes,
    const int n_threads, const int* n_devices_network, const int n_itr,
    std::vector<std::tuple<std::vector<SERP_Hst>, std::vector<SERP_Hst>, int>>& device_partitions,
    const std::vector<std::unordered_map<int, std::unordered_map<int, int>>*>& root_mapping) {
    int n_devices = device_partitions.size();

    if (node_id == ROOT) {
        std::cout << "\nExpectation Maximization (EM) in parallel ..." << std::endl;
    }


    //-----------------------------------------------------------------------//
    // Initate host-side click model.                                        //
    //-----------------------------------------------------------------------//

    // Initiate a host-side click model for each device.
    ClickModel_Hst* cm_hosts[n_devices];
    for (int device_id = 0; device_id < n_devices; device_id++) {
        // Initialize the click model.
        cm_hosts[device_id] = create_cm_host(model_type);
        // Print a confirmation message on the first device of the root node.
        if (node_id == ROOT && device_id == 0) {
            cm_hosts[device_id]->say_hello();
        }
    }


    //-----------------------------------------------------------------------//
    // Assign queries to CPU threads.                                        //
    //-----------------------------------------------------------------------//

    std::vector<std::vector<int>> thread_start_idx(n_devices);
    int n_queries_total = std::accumulate(device_partitions.begin(), device_partitions.end(), 0,
        [](int sum, const std::tuple<std::vector<SERP_Hst>, std::vector<SERP_Hst>, int>& partition) {
           return sum + std::get<0>(partition).size();});
    int available_threads = n_threads;
    int device_id = 0;
    // Allocate CPU threads to each device based on the number of assigned
    // queries.
    while (available_threads > 0) {
        int n_queries = std::get<0>(device_partitions[device_id]).size();
        float ratio = (float) n_queries / (float) n_queries_total;
        int n_threads_device = std::round(ratio * n_threads);

        if (thread_start_idx[device_id].size() < 1) {
            thread_start_idx[device_id].push_back(-1);
            available_threads--;
        }
        else if (thread_start_idx[device_id].size() < n_threads_device) {
            thread_start_idx[device_id].push_back(-1);
            available_threads--;
        }

        device_id = (device_id + 1) % n_devices;
    }

    // Determine the number of queries per thread for each device.
    for (int did = 0; did < n_devices; did++) {
        int n_threads_device = thread_start_idx[did].size();
        int n_queries = std::get<0>(device_partitions[did]).size();
        int stride = (float) n_queries / (float) n_threads_device;

        // Use only a single thread if there are not enough queries to
        // distribute over all threads.
        if (n_queries < n_threads_device) {
            thread_start_idx[did].resize(1);
            thread_start_idx[did][0] = 0;
        }
        else {
            int n_unused_threads = 0;
            // Determine the starting query index for each thread.
            for (int tid = 0; tid < n_threads_device; tid++) {
                int curr_query = std::get<0>(device_partitions[did])[tid * stride].get_query();
                int start_index = tid * stride;

                // Check if the current query has been assigned to a previous
                // thread. If so, assign the next query to the current thread.
                bool duplicate = false;
                if (tid - 1 > 0) {
                    int prev_query = std::get<0>(device_partitions[did])[thread_start_idx[did][tid - 1]].get_query();
                    if (curr_query == prev_query) {
                        duplicate = true;
                        for (int i = start_index; i < n_queries; i++) {
                            // If a query is found that is not assigned to a
                            // previous thread, assign it to the current thread.
                            if (std::get<0>(device_partitions[did])[i].get_query() != curr_query) {
                                // Reassign the current query and starting index.
                                duplicate = false;
                                start_index = i;
                                curr_query = std::get<0>(device_partitions[did])[i].get_query();
                                break;
                            }
                        }
                        if (duplicate) {
                            // If no query is found that is not assigned to a
                            // previous thread, increase the number of unused
                            // threads. These will be removed later.
                            n_unused_threads++;
                        }
                        break;
                    }
                }

                // Look behind in the partition to find the start of the current
                // set of similar queries.
                for (int i = start_index; i >= 0; i--) {
                    if (std::get<0>(device_partitions[did])[i].get_query() != curr_query) {
                        thread_start_idx[did][tid] = i + 1;
                        break;
                    }
                    else if (i == 0) {
                        thread_start_idx[did][tid] = 0;
                    }
                }
            }

            // Remove unused threads from the current partition and assign them
            // to the next partition, if possible.
            if (n_unused_threads > 0) {
                thread_start_idx[did].resize(n_threads_device - n_unused_threads);
                // Add the removed threads to the next partition.
                if (did < n_devices - 1) {
                    thread_start_idx[did + 1].resize(thread_start_idx[did + 1].size() + n_unused_threads, -1);
                }
            }
        }
    }


    //-----------------------------------------------------------------------//
    // Allocate memory.                                                      //
    //-----------------------------------------------------------------------//

    auto h2d_init_start_time = std::chrono::high_resolution_clock::now();

    // Allocate memory on the device.
    SearchResult_Dev* dataset_dev[n_devices];
    size_t fmem_dev[n_devices * 2];
    for (int device_id = 0; device_id < n_devices; device_id++) {
        CUDA_CHECK(hipSetDevice(device_id));

        // Retrieve avaliable memory in bytes.
        size_t fmem, tmem;
        get_device_memory(device_id, fmem, tmem, 1);
        fmem_dev[device_id * 2] = 0; // Memory in use.
        fmem_dev[device_id * 2 + 1] = fmem; // Total available memory.

        // Convert the host-side dataset to a smaller device-side dataset.
        std::vector<SearchResult_Dev> dataset_dev_tmp;
        convert_to_device(std::get<0>(device_partitions[device_id]), dataset_dev_tmp);

        // Check whether the current device has enough free memory available.
        double dataset_size = dataset_dev_tmp.size() * sizeof(SearchResult_Dev);
        if (dataset_size * 1.001 > fmem) {
            std::cout << "Error: Insufficient GPU memory!\n\tAllocating dataset requires an additional " <<
            (dataset_size - fmem_dev[device_id * 2 + 1]) / 1e6 << " MB of GPU memory." << std::endl;
            mpi_abort(-1);
        }

        // Allocate memory for the dataset on the current device.
        CUDA_CHECK(hipMalloc(&dataset_dev[device_id], dataset_size));
        CUDA_CHECK(hipMemcpy(dataset_dev[device_id], dataset_dev_tmp.data(),
                              dataset_size, hipMemcpyHostToDevice));
        dataset_dev_tmp.clear();

        fmem_dev[device_id * 2] += dataset_size;

        // Allocate memory for the query dependent parameters on both the current device and host.
        cm_hosts[device_id]->init_parameters(device_partitions[device_id], fmem_dev[device_id * 2 + 1] - fmem_dev[device_id * 2]);
        fmem_dev[device_id * 2] += cm_hosts[device_id]->get_memory_usage();

        // Show memory usage.
        size_t fmem_new, tmem_new;
        get_device_memory(device_id, fmem_new, tmem_new, 1);

        std::cout << "(" << node_id << ", " << device_id << "), expected GPU memory usage = " <<
        fmem_dev[device_id * 2] / 1e6 << "/" << fmem_dev[device_id * 2 + 1] / 1e6 << " MB (" <<
        (int) ((float) fmem_dev[device_id * 2] / (float) fmem_dev[device_id * 2 + 1] * 100) <<
        "%)\n\tmeasured GPU memory usage = " << (fmem - fmem_new) / 1e6 << "/" <<
        fmem_dev[device_id * 2 + 1] / 1e6 << " MB (" <<
        (int) ((float) (fmem - fmem_new) / (float) fmem_dev[device_id * 2 + 1] * 100) << "%)" << std::endl;
    }

    Communicate::barrier();

    auto h2d_init_stop_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> h2d_init_elapsed_time = h2d_init_stop_time - h2d_init_start_time;


    //-----------------------------------------------------------------------//
    // Initate device-side click model.                                      //
    //-----------------------------------------------------------------------//

    // Initialize the device-side click model.
    for (int device_id = 0; device_id < n_devices; device_id++) {
        CUDA_CHECK(hipSetDevice(device_id));

        // Retrieve the device-side click model parameter arrays and their sizes.
        Param** parameter_references;
        int* parameter_sizes;
        cm_hosts[device_id]->get_device_references(parameter_references, parameter_sizes);

        // Launch the click model initialization kernel.
        Kernel::initialize<<<1, 1>>>(model_type, node_id, device_id, parameter_references, parameter_sizes);

        CUDA_CHECK(hipDeviceSynchronize());
    }


    //-----------------------------------------------------------------------//
    // Estimate CM parameters n_itr times.                                   //
    //-----------------------------------------------------------------------//

    // Initiate CUDA event timers.
    hipEvent_t start_events[n_devices], end_events[n_devices];
    double avg_time_comp{0}, avg_time_update{0};
    std::chrono::duration<double> tot_time_em{0}, avg_time_itr{0}, avg_time_sync{0}, avg_time_h2d{0}, avg_time_d2h{0};
    for (int dev = 0; dev < n_devices; dev++) {
        CUDA_CHECK(hipSetDevice(dev));
        CUDA_CHECK(hipEventCreate(&start_events[dev]));
        CUDA_CHECK(hipEventCreate(&end_events[dev]));
    }

    // Get kernel dimensions.
    int kernel_dims[n_devices * 2];
    for (int did = 0; did < n_devices; did++) {
        int n_queries = std::get<0>(device_partitions[did]).size(); // Number of non-unique queries in the dataset.
        // Number of threads per block.
        int block_size = BLOCK_SIZE;
        // Calculate the number of blocks in which the array size can be split
        // up. (block_size - 1) is used to ensure that there won't be an
        // insufficient amount of blocks.
        kernel_dims[did * 2] = (n_queries + (block_size - 1)) / block_size;
        kernel_dims[did * 2 + 1] = block_size;

        std::cout << "(" << node_id << ", " << did << "), kernel dimensions = <<<" << kernel_dims[did * 2] << ", " << kernel_dims[did * 2 + 1] << ">>>" << std::endl;
    }

    if (node_id == ROOT) {
        std::cout << "\nStarting " << n_itr << " EM parameter estimation iterations..." << std::endl;
    }

    // Perform n_itr Expectation-Maximization iterations.
    for (int itr = 0; itr < n_itr; itr++) {

        //-------------------------------------------------------------------//
        // Launch parameter estimation kernel.                               //
        //-------------------------------------------------------------------//

        auto em_itr_start_time = std::chrono::high_resolution_clock::now();

        for (int device_id = 0; device_id < n_devices; device_id++) {

            CUDA_CHECK(hipSetDevice(device_id));

            int grid_size = kernel_dims[device_id * 2];
            int block_size = kernel_dims[device_id * 2 + 1];
            int dataset_size = std::get<0>(device_partitions[device_id]).size();

            CUDA_CHECK(hipEventRecord(start_events[device_id], 0));
            Kernel::em_training<<<grid_size, block_size>>>(dataset_dev[device_id], dataset_size);
            CUDA_CHECK(hipEventRecord(end_events[device_id], 0));
        }

        for (int device_id = 0; device_id < n_devices; device_id++) {
            CUDA_CHECK(hipSetDevice(device_id));
            CUDA_CHECK(hipDeviceSynchronize());
        }

        double em_comp_elapsed_time{0};
        for (int device_id = 0; device_id < n_devices; device_id++) {
            CUDA_CHECK(hipSetDevice(device_id));
            float time_ms;
            CUDA_CHECK(hipEventElapsedTime(&time_ms, start_events[device_id], end_events[device_id]));
            em_comp_elapsed_time += (double) ((time_ms / 1000.f) / ((double) n_devices));
        }

        avg_time_comp += em_comp_elapsed_time / ((double) n_itr);


        //-------------------------------------------------------------------//
        // Wipe previous parameter results.                                  //
        //-------------------------------------------------------------------//

        for (int device_id = 0; device_id < n_devices; device_id++) {
            CUDA_CHECK(hipSetDevice(device_id));

            auto h2d_start_time = std::chrono::high_resolution_clock::now();

            cm_hosts[device_id]->reset_parameters();

            auto h2d_stop_time = std::chrono::high_resolution_clock::now();
            avg_time_h2d += (h2d_stop_time - h2d_start_time) / n_itr;
        }

        for (int device_id = 0; device_id < n_devices; device_id++) {
            CUDA_CHECK(hipSetDevice(device_id));
            CUDA_CHECK(hipDeviceSynchronize());
        }


        //-------------------------------------------------------------------//
        // Launch parameter update kernel.                                   //
        //-------------------------------------------------------------------//

        for (int device_id = 0; device_id < n_devices; device_id++) {
            CUDA_CHECK(hipSetDevice(device_id));

            int grid_size = kernel_dims[device_id * 2];
            int block_size = kernel_dims[device_id * 2 + 1];
            int dataset_size = std::get<0>(device_partitions[device_id]).size();

            CUDA_CHECK(hipEventRecord(start_events[device_id], 0));
            Kernel::update<<<grid_size, block_size>>>(dataset_dev[device_id], dataset_size);
            // cm_hosts[device_id]->update_parameters_on_host(thread_start_idx[device_id], std::get<0>(device_partitions[device_id]));

            CUDA_CHECK(hipEventRecord(end_events[device_id], 0));
        }

        for (int device_id = 0; device_id < n_devices; device_id++) {
            CUDA_CHECK(hipSetDevice(device_id));
            CUDA_CHECK(hipDeviceSynchronize());
        }

        double em_update_elapsed_time{0};
        for (int device_id = 0; device_id < n_devices; device_id++) {
            CUDA_CHECK(hipSetDevice(device_id));
            float time_ms;
            CUDA_CHECK(hipEventElapsedTime(&time_ms, start_events[device_id], end_events[device_id]));
            em_update_elapsed_time += (double) ((time_ms / 1000.f) / ((double) n_devices));
        }
        avg_time_update += em_update_elapsed_time / n_itr;


        //-------------------------------------------------------------------//
        // Synchronize parameters across the nodes and devices.              //
        //-------------------------------------------------------------------//

        auto em_sync_itr_start_time = std::chrono::high_resolution_clock::now();

        std::vector<std::vector<std::vector<Param>>> public_parameters(n_devices); // Device ID -> Parameter type -> Parameters.

        // Retrieve all types of public parameters from each device.
        for (int device_id = 0; device_id < n_devices; device_id++) {
            CUDA_CHECK(hipSetDevice(device_id));

            auto d2h_start_time = std::chrono::high_resolution_clock::now();

            cm_hosts[device_id]->transfer_parameters(PUBLIC, D2H);

            auto d2h_stop_time = std::chrono::high_resolution_clock::now();
            avg_time_d2h += (d2h_stop_time - d2h_start_time) / n_itr;

            cm_hosts[device_id]->get_parameters(public_parameters[device_id], PUBLIC);
        }

        // Synchronize the parameters local to this device before synchronizing
        // with the parameters from other nodes.
        cm_hosts[0]->sync_parameters(public_parameters);

        // Send this node's synchronized public device parameters to all other
        // nodes in the network.
        std::vector<std::vector<std::vector<Param>>> network_parameters(n_nodes,
            std::vector<std::vector<Param>>(public_parameters.size(),
            std::vector<Param>(public_parameters[0][0].size()))); // Node ID -> Parameter type -> Parameters.
        Communicate::exchange_parameters(network_parameters, public_parameters[0], n_nodes, node_id);

        // Sychronize the public parameters received from other nodes.
        cm_hosts[0]->sync_parameters(network_parameters);

        // Move all types of synchronized public parameters back to each device.
        for (int device_id = 0; device_id < n_devices; device_id++) {
            cm_hosts[device_id]->set_parameters(network_parameters[0], PUBLIC);

            auto h2d_start_time = std::chrono::high_resolution_clock::now();

            cm_hosts[device_id]->transfer_parameters(PUBLIC, H2D);

            auto h2d_stop_time = std::chrono::high_resolution_clock::now();
            avg_time_h2d += (h2d_stop_time - h2d_start_time) / n_itr;
        }

        auto em_sync_itr_stop_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> em_sync_elapsed_time = em_sync_itr_stop_time - em_sync_itr_start_time;
        avg_time_sync += em_sync_elapsed_time / n_itr;

        auto em_itr_stop_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> em_itr_elapsed_time = em_itr_stop_time - em_itr_start_time;
        avg_time_itr += em_itr_elapsed_time / n_itr;
        tot_time_em += em_itr_elapsed_time;


        // Show metrics on the root node.
        if (node_id == ROOT) {
            int itr_len = std::to_string(n_itr).length();
            std::cout << "Itr: " << std::left << std::setw(itr_len) << itr <<
            " Itr-time: " << std::left << std::setw(10) << em_itr_elapsed_time.count() <<
            " Itr-EM_COMP: " << std::left << std::setw(11) << em_comp_elapsed_time <<
            " Itr-EM_UPDATE: " << std::left << std::setw(10) << em_update_elapsed_time <<
            " Itr-Sync: " << std::left << std::setw(12) << em_sync_elapsed_time.count() << std::endl;
        }
    }

    // Destroy CUDA timer events.
    for (int device_id = 0; device_id < n_devices; device_id++) {
        CUDA_CHECK(hipEventDestroy(start_events[device_id]));
        CUDA_CHECK(hipEventDestroy(end_events[device_id]));
    }


    //-----------------------------------------------------------------------//
    // Copy trained (partial) click model from the device to the host.       //
    //-----------------------------------------------------------------------//

    for (int device_id = 0; device_id < n_devices; device_id++) {
        CUDA_CHECK(hipSetDevice(device_id));

        // Ensure that all kernels have finished their execution.
        CUDA_CHECK(hipDeviceSynchronize());

        // Copy device-side parameters to host.
        cm_hosts[device_id]->transfer_parameters(ALL, D2H);
    }


    //-----------------------------------------------------------------------//
    // Evaluate CM using log-likelihood and perplexity on each node.         //
    //-----------------------------------------------------------------------//

    auto em_eval_start_time = std::chrono::high_resolution_clock::now();

    // Calculate node-local log-likelihood and perplexity.
    std::map<int, std::array<float, 2>> llh_device;
    std::map<int, Perplexity> ppl_device;
    for (int device_id = 0; device_id < n_devices; device_id++) {
        // Compute the log-likelihood.
        LogLikelihood llh(cm_hosts[device_id]);
        float llh_val = llh.evaluate(std::get<1>(device_partitions[device_id]));
        int task_size = std::get<1>(device_partitions[device_id]).size();
        std::array<float, 2> temp_arr{llh_val, static_cast<float>(task_size)};
        llh_device[device_id] = temp_arr;

        // Compute the perplexity.
        Perplexity ppl;
        ppl.evaluate(cm_hosts[device_id], std::get<1>(device_partitions[device_id]));
        ppl_device[device_id] = ppl;
    }

    // Gather the log-likelihood and perplexity of all nodes on the root node.
    Communicate::gather_evaluations(llh_device, ppl_device, n_nodes, node_id, n_devices_network);

    auto em_eval_stop_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> em_eval_elapsed_time = em_eval_stop_time - em_eval_start_time;


    //-----------------------------------------------------------------------//
    // Free allocated device-side memory.                                    //
    //-----------------------------------------------------------------------//

    for (int device_id = 0; device_id < n_devices; device_id++) {
        CUDA_CHECK(hipSetDevice(device_id));
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipFree(dataset_dev[device_id]));
        cm_hosts[device_id]->destroy_parameters();
    }


    //-----------------------------------------------------------------------//
    // Create complete click model.                                          //
    //-----------------------------------------------------------------------//
    // Gather public and private parameters and log-likelihood and perplexity
    // from all nodes and devices.

    // Create complete click model on root node.
    if (node_id == ROOT) {
        // TODO: Gather public and private parameters from all nodes and devices.
        // TODO: The root-side param hash-map (root_mapping) can be used to lookup specific qd-pairs.
        // TODO: Gather log-likelihood and perplexity from all nodes and devices.
    }


    //-----------------------------------------------------------------------//
    // Show metrics.                                                         //
    //-----------------------------------------------------------------------//

    // Show metrics on the root node.
    if (node_id == ROOT) {
        // Compute the total log-likelihood.
        float total_llh_sum = 0.0;
        float total_llh_sessions = 0.0;
        std::for_each(llh_device.begin(), llh_device.end(), [&] (std::pair<const int, std::array<float, 2>>& llh_task) {
            total_llh_sum += llh_task.second[0];
            total_llh_sessions += llh_task.second[1];
        });

        std::cout << "\nTotal Log likelihood is: " << total_llh_sum / total_llh_sessions << std::endl;

        float total_task_size{0.0};
        std::array<float, 10> temp_task_rank_perplexities{0.0};

        // Compute perplexity at every rank.
        for (auto const& itr: ppl_device){
            total_task_size += itr.second.task_size;
            for (int j{0}; j < 10; j++){
                temp_task_rank_perplexities[j] += itr.second.task_rank_perplexities[j];
            }
        }

        std::array<float, 10> ppl_per_rank{};
        float ppl_value;
        for (int i{0}; i < 10; i++){
            ppl_per_rank[i] = std::pow(2, (-1 * temp_task_rank_perplexities[i])/total_task_size);
            std::cout << "Perplexity at rank " << i << " is: " << ppl_per_rank[i] << std::endl;
        }
        ppl_value = std::accumulate(ppl_per_rank.begin(), ppl_per_rank.end(), 0.0) / 10.0;
        std::cout << "Perplexity is: " << ppl_value << std::endl;

        // Show the timing measurements of the EM algorithm.
        std::cout << "\nHost to Device dataset transfer time: " << h2d_init_elapsed_time.count() <<
        "\nAverage Host to Device parameter transfer time: " << avg_time_h2d.count() <<
        "\nAverage Device to Host parameter transfer time: " << avg_time_d2h.count() << std::endl;

        std::cout << "\nAverage time per iteration: " << avg_time_itr.count() <<
        "\nAverage time per computation in each Iteration: " << avg_time_comp <<
        "\nAverage time per update in each Iteration: " << avg_time_update <<
        "\nAverage time per synchronization in each Iteration: " << avg_time_sync.count() <<
        "\nTotal time of training: " << tot_time_em.count() <<
        "\nEvaluation time: " << em_eval_elapsed_time.count() << std::endl;
    }

    // Destroy all allocations on all available devices as part of the shutdown
    // procedure.
    for (int device_id = 0; device_id < n_devices; device_id++) {
        CUDA_CHECK(hipSetDevice(device_id));
        CUDA_CHECK(hipDeviceReset());
    }
}